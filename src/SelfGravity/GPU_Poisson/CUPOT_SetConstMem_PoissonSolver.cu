#include "Macro.h"
#include "CUPOT.h"

#if ( defined GRAVITY  &&  defined GPU )


__constant__ real c_Mp[3];
__constant__ real c_Mm[3];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_SetConstMem_PoissonSolver
// Description :  Set the constant memory used by CUPOT_PoissonSolver_SOR_10to14cube()
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by CUAPI_Set_Default_GPU_Parameter()
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
__host__
int CUPOT_SetConstMem_PoissonSolver()
{

   const real h_Mp[3] = { -3.0/32.0, +30.0/32.0, +5.0/32.0 };
   const real h_Mm[3] = { +5.0/32.0, +30.0/32.0, -3.0/32.0 };

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_Mp), h_Mp, 3*sizeof(real), 0, hipMemcpyHostToDevice)  )
   return -1;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_Mm), h_Mm, 3*sizeof(real), 0, hipMemcpyHostToDevice)  )
   return -1;

   return 0;

} // FUNCTION : CUPOT_SetConstMem_PoissonSolver



#endif // #if ( defined GRAVITY  &&  defined GPU )
