#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUFLU.h"

#if ( defined GPU  &&  MODEL == ELBDM  &&  WAVE_SCHEME == WAVE_FD )



// useful macros
#define to1D1(z,y,x) ( __umul24(z, FLU_NXT*FLU_NXT) + __umul24(y, FLU_NXT) + x )
#define to1D2(z,y,x) ( __umul24(z-FLU_GHOST_SIZE, PS2*PS2) + __umul24(y-FLU_GHOST_SIZE, PS2) + x-FLU_GHOST_SIZE )

#ifdef LAPLACIAN_4TH
#  define LAP_GHOST  2
#  define LAP1(In,i)    (  real(1.0/ 12.0)*( - In[i-2] + (real)16.0*In[i-1] - (real)30.0*In[i  ] \
                                             - In[i+2] + (real)16.0*In[i+1] )  )
#  define LAP2(In,i)    (  real(1.0/144.0)*( + In[i-4] - (real)32.0*In[i-3] + (real)316.0*In[i-2] - (real)992.0*In[i-1] \
                                             + In[i+4] - (real)32.0*In[i+3] + (real)316.0*In[i+2] - (real)992.0*In[i+1] \
                                             +  (real)1414.0*In[i  ] )  )
#  ifndef CONSERVE_MASS
#  define LAP3(In,i)    (  real(1.0/1728.0)* \
       (  -In[i-6] + (real)48*In[i-5] - (real)858*In[i-4] + (real)7024*In[i-3] - (real)27279*In[i-2] + (real)58464*In[i-1] \
          -In[i+6] + (real)48*In[i+5] - (real)858*In[i+4] + (real)7024*In[i+3] - (real)27279*In[i+2] + (real)58464*In[i+1] \
          - (real)74796*In[i  ] )  )
#  endif

#else // #ifdef LAPLACIAN_4TH

#  define LAP_GHOST     1
#  define LAP1(In,i)    ( + In[i-1] - (real)2.0*In[i  ] + In[i+1] )
#  define LAP2(In,i)    ( + In[i-2] - (real)4.0*In[i-1] + (real)6.0*In[i  ] - (real)4.0*In[i+1] + In[i+2] )
#  ifndef CONSERVE_MASS
#  define LAP3(In,i)    ( + In[i-3] - (real)6.0*In[i-2] + (real)15.0*In[i-1] - (real)20.0*In[i  ] \
                          + In[i+3] - (real)6.0*In[i+2] + (real)15.0*In[i+1] )

#  endif

#endif // #ifdef LAPLACIAN_4TH ... else ...


static __device__ void CUFLU_Advance( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                                      real g_Fluid_Out[][FLU_NOUT][ CUBE(PS2) ],
                                      real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                      const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                      const real Taylor3_Coeff, const uint j_gap, const uint k_gap,
                                      real s_In[][FLU_BLOCK_SIZE_Y][FLU_NXT], real s_Half[][FLU_BLOCK_SIZE_Y][FLU_NXT],
                                      real s_Flux[][PS2+1], const bool FinalOut, const int XYZ, const real MinDens );




//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_ELBDMSolver_FD
// Description :  GPU ELBDM kinematic solver based on expanding the propagator to 3rd order
//
// Note        :  1. The three-dimensional evolution is achieved by applying x, y, and z operators successively.
//                   Since these operators commute, the order of applying them are irrelevant.
//                   --> Input pamameter "XYZ" is actually useless
//                   --> Nevertheless, the symmetry in different directions will be broken if CONSERVE_MASS is on
//                2. The implementation is very similar to the function " CUFLU_FluidSolver_RTVD"
//                4. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//
// Parameter   :  g_Fluid_In     : Global memory array storing the input variables
//                g_Fluid_Out    : Global memory array to store the output variables
//                g_Flux         : Global memory array to store the output fluxes (useful only if StoreFlux == true)
//                dt             : Time interval to advance solution
//                _dh            : 1 / grid size
//                Eta            : Particle mass / Planck constant
//                StoreFlux      : true --> store the coarse-fine fluxes
//                                      --> useful only if CONSERVE_MASS is defined
//                Taylor3_Coeff  : Coefficient in front of the third term in the Taylor expansion
//                XYZ            : true  : x->y->z ( forward sweep)
//                                 false : z->y->x (backward sweep)
//                                 --> Meaningless if CONSERVE_MASS is off since the operators along different directions
//                                     commute
//                                 --> Meaningful if CONSERVE_MASS is on, in which the symmetry along different directions
//                                     are broken ...
//                MinDens        : Minimum allowed density
//-------------------------------------------------------------------------------------------------------
__global__ void CUFLU_ELBDMSolver_FD( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                                      real g_Fluid_Out[][FLU_NOUT][ CUBE(PS2) ],
                                      real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                      const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                      const real Taylor3_Coeff, const bool XYZ, const real MinDens )
{

   __shared__ real s_In  [FLU_NIN][FLU_BLOCK_SIZE_Y][FLU_NXT];
#  ifdef CONSERVE_MASS
   __shared__ real s_Half[FLU_NIN][FLU_BLOCK_SIZE_Y][FLU_NXT];
   __shared__ real s_Flux[FLU_BLOCK_SIZE_Y][PS2+1];
#  else
   real (*s_Half)[FLU_BLOCK_SIZE_Y][FLU_NXT] = NULL;  // useless if CONSERVE_MASS is off
   real (*s_Flux)[PS2+1]                     = NULL;  // useless if CONSERVE_MASS is off
#  endif

   if ( XYZ )
   {
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                                  0,              0, s_In, s_Half, s_Flux, false, 0, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE,              0, s_In, s_Half, s_Flux, false, 3, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE, FLU_GHOST_SIZE, s_In, s_Half, s_Flux,  true, 6, MinDens );
   }

   else
   {
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                                  0,              0, s_In, s_Half, s_Flux, false, 6, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                                  0, FLU_GHOST_SIZE, s_In, s_Half, s_Flux, false, 3, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE, FLU_GHOST_SIZE, s_In, s_Half, s_Flux,  true, 0, MinDens );
   }

} // FUNCTION : CUFLU_ELBDMSolver_FD



//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_Advance
// Description :  Use GPU to advance solutions by one time-step
//
// Note        :  1. Based on expanding the kinematic propagator to 3rd order
//                2. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                3. The direction of the one dimensional sweep is determined by the input parameter "XYZ"
//
// Parameter   :  g_Fluid_In     : Global memory array storing the input variables
//                g_Fluid_Out    : Global memory array to store the output variables
//                g_Flux         : Global memory array to store the output fluxes (useful only if StoreFlux == true)
//                dt             : Time interval to advance solution
//                _dh            : 1 / grid size
//                Eta            : Particle mass / Planck constant
//                StoreFlux      : true --> store the coarse-fine fluxes
//                                   --> useful only if CONSERVE_MASS is defined
//                Taylor3_Coeff  : Coefficient in front of the third term in the Taylor expansion
//                j_gap          : Number of useless grids on each side in the j direction (j may not be equal to y)
//                k_gap          : Number of useless grids on each side in the k direction (k mya not be equal to z)
//                s_In           : Shared memory array to store the input data
//                s_Half         : Shared memory array to store the half-step solution
//                s_Flux         : Shared memory array to store the boundary fluxes
//                FinalOut       : true --> store the updated data to g_Fluid_Out
//                XYZ            : 0 : Update the solution in the x direction
//                                 3 : Update the solution in the y direction
//                                 6 : Update the solution in the z direction
//                                 --> This parameter is also used to determine the place to store the output fluxes
//                MinDens        : Minimum allowed density
//-------------------------------------------------------------------------------------------------------
__device__ void CUFLU_Advance( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                               real g_Fluid_Out[][FLU_NOUT][ CUBE(PS2) ],
                               real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                               const real dt, const real _dh, const real Eta, const bool StoreFlux, const real Taylor3_Coeff,
                               const uint j_gap, const uint k_gap, real s_In[][FLU_BLOCK_SIZE_Y][FLU_NXT],
                               real s_Half[][FLU_BLOCK_SIZE_Y][FLU_NXT], real s_Flux[][PS2+1], const bool FinalOut,
                               const int XYZ, const real MinDens )
{

   const real _Eta         = (real)1.0/Eta;
   const real dT           = (real)0.5*dt*_Eta;
   const real _Eta2_dh     = (real)0.5*_dh*_Eta;
   const real Coeff1       = dT*_dh*_dh;
#  ifdef CONSERVE_MASS
   const real Coeff2       = Taylor3_Coeff*SQR(Coeff1);
#  else
   const real Coeff2       = (real)0.5*SQR(Coeff1);
   const real Coeff3       = Taylor3_Coeff*CUBE(Coeff1);
#  endif

   const uint bx           = blockIdx.x;
   const uint tx           = threadIdx.x;
   const uint ty           = threadIdx.y;
   const uint tid          = __umul24(ty,FLU_BLOCK_SIZE_X) + tx;
   const uint size_j       = FLU_NXT - (j_gap<<1);
   const uint size_k       = FLU_NXT - (k_gap<<1);
   const uint NColumnTotal = __umul24( size_j, size_k );    // total number of data columns to be updated
   const uint i            = tx + FLU_GHOST_SIZE;           // (i,j,k): array indices used in g_Fluid_In
   const uint j_end       = FLU_NXT - j_gap;
         uint j           = j_gap + ty%size_j;
         uint k           = k_gap + ty/size_j;
         uint Column0     = 0;                              // the total number of columns that have been updated
         uint NColumnOnce = MIN( NColumnTotal, FLU_BLOCK_SIZE_Y );

   double Amp_New;            // use double precision to reduce the round-off error in the mass conservation
   real   Re_Old, Im_Old, Re_New, Im_New;
   uint   Idx1, Idx2, Idx3, delta_k;

#  ifdef CONSERVE_MASS
   const uint NThread     = FLU_BLOCK_SIZE_X*FLU_BLOCK_SIZE_Y;
   const uint NHalf       = FLU_NXT - 4*LAP_GHOST;
   const real dT_dh2      = dT*_dh*_dh;
   const uint txp         = tx + 1;

   double Amp_Old, Amp_Corr;  // use double precision to reduce the round-off error in the mass conservation
   real   R, I, dR, dI;
   uint   Idx;
   uint   si, sj;                                           // array indices used in the shared memory array
   uint   f, fp1;                                           // array indices used in the s_Flux array
#  ifdef LAPLACIAN_4TH
   uint   fm1, fp2;
#  endif
#  endif // #ifdef CONSERVE_MASS


// determine the array indices for loading the ghost-zone data
   bool LoadGhost = false;                                  // true --> load the ghost-zone data
   uint LoadGhost_i;
   int  LoadGhost_di, LoadGhost_dIdx1;

   if ( tx < 2*FLU_GHOST_SIZE )
   {
      LoadGhost = true;

      if ( tx < FLU_GHOST_SIZE )    LoadGhost_di = -FLU_GHOST_SIZE;
      else                          LoadGhost_di = -FLU_GHOST_SIZE + PS2;

      switch ( XYZ )
      {
         case 0:  LoadGhost_dIdx1 = LoadGhost_di;                                break;
         case 3:  LoadGhost_dIdx1 = __mul24( LoadGhost_di, FLU_NXT );            break;
         case 6:  LoadGhost_dIdx1 = __mul24( LoadGhost_di, FLU_NXT*FLU_NXT );    break;
      }

      LoadGhost_i = (int)i + LoadGhost_di;
   } // if ( tx < 2*FLU_GHOST_SIZE )


// loop over all data columns
   while ( Column0 < NColumnTotal )
   {
//    1. load data into shared memory
      if ( tid < NColumnOnce*PS2 )
      {
//       1.1 determine the array indices for loading global memory data along different directions
         switch ( XYZ )
         {
            case 0:  Idx1 = to1D1( k, j, i );    break;
            case 3:  Idx1 = to1D1( k, i, j );    break;
            case 6:  Idx1 = to1D1( i, k, j );    break;
         }

//       1.2 load the interior data into shared memory
         Re_Old = g_Fluid_In[bx][0][Idx1];
         Im_Old = g_Fluid_In[bx][1][Idx1];

         s_In[0][ty][i] = Re_Old;
         s_In[1][ty][i] = Im_Old;

//       1.3 load the ghost-zone data into shared memory
         if ( LoadGhost )
         {
            s_In[0][ty][LoadGhost_i] = g_Fluid_In[bx][0][ (int)Idx1 + LoadGhost_dIdx1 ];
            s_In[1][ty][LoadGhost_i] = g_Fluid_In[bx][1][ (int)Idx1 + LoadGhost_dIdx1 ];
         }
      } // if ( tid < NColumnOnce*PS2 )

      __syncthreads();


#     ifdef CONSERVE_MASS


//    2. half-step solution
      Idx = tid;
      while ( Idx < NColumnOnce*NHalf )
      {
         si = Idx % NHalf + 2*LAP_GHOST;
         sj = Idx / NHalf;

         s_Half[0][sj][si] = s_In[0][sj][si] - (real)0.5*Coeff1*LAP1( s_In[1][sj], si ) - Coeff2*LAP2( s_In[0][sj], si );
         s_Half[1][sj][si] = s_In[1][sj][si] + (real)0.5*Coeff1*LAP1( s_In[0][sj], si ) - Coeff2*LAP2( s_In[1][sj], si );

         Idx += NThread;
      } // while ( Idx < NColumnOnce*NHalf )

      __syncthreads();


//    3. calculate the face-center fluxes (the coefficient _dh has been absorted into the constant dT_dh2)
      Idx = tid;
      while ( Idx < NColumnOnce*(PS2+1) )
      {
         si  = Idx % (PS2+1);
         sj  = Idx / (PS2+1);
         f   = si + FLU_GHOST_SIZE - 1;
         fp1 = f + 1;

#        ifdef LAPLACIAN_4TH
         fm1 = f - 1;
         fp2 = f + 2;

         R  = real(1./28.)*( -s_Half[0][sj][fm1]+(real)15*s_Half[0][sj][f]+(real)15*s_Half[0][sj][fp1]-s_Half[0][sj][fp2] );
         I  = real(1./28.)*( -s_Half[1][sj][fm1]+(real)15*s_Half[1][sj][f]+(real)15*s_Half[1][sj][fp1]-s_Half[1][sj][fp2] );
         dR = real(1./12.)*( +s_Half[0][sj][fm1]-(real)15*s_Half[0][sj][f]+(real)15*s_Half[0][sj][fp1]-s_Half[0][sj][fp2] );
         dI = real(1./12.)*( +s_Half[1][sj][fm1]-(real)15*s_Half[1][sj][f]+(real)15*s_Half[1][sj][fp1]-s_Half[1][sj][fp2] );

#        else

         R  = real(0.5)*( + s_Half[0][sj][f] + s_Half[0][sj][fp1] );
         I  = real(0.5)*( + s_Half[1][sj][f] + s_Half[1][sj][fp1] );
         dR =           ( - s_Half[0][sj][f] + s_Half[0][sj][fp1] );
         dI =           ( - s_Half[1][sj][f] + s_Half[1][sj][fp1] );
#        endif

         s_Flux[sj][si] = (real)2.0*( R*dI - I*dR );

         Idx += NThread;
      } // while ( Idx < NColumnOnce*(PS2+1) )

      __syncthreads();


//    4a. full-step solution (equivalent to the 3rd-order Taylor expansion)
      if ( tid < NColumnOnce*PS2 )
      {
         Re_New   = Re_Old - Coeff1*LAP1( s_Half[1][ty], i );
         Im_New   = Im_Old + Coeff1*LAP1( s_Half[0][ty], i );

         Amp_Old  = SQR( Re_Old ) + SQR( Im_Old );
         Amp_New  = SQR( Re_New ) + SQR( Im_New );
         Amp_Corr = Amp_Old - dT_dh2*( s_Flux[ty][txp] - s_Flux[ty][tx] );

//       be careful about the negative density and the vacuum (where we might have Amp_New == 0.0)
//       if ( Amp_Corr > (real)0.0  &&  Amp_New > (real)0.0 )
         if ( Amp_Corr >       0.0  &&  Amp_New >       0.0 )
         {
            /*
            Re_New *= SQRT( Amp_Corr / Amp_New );
            Im_New *= SQRT( Amp_Corr / Amp_New );
            */
            Re_New *= sqrt( Amp_Corr / Amp_New );  // use double precision to improve the mass conservation further
            Im_New *= sqrt( Amp_Corr / Amp_New );
            Amp_New = Amp_Corr;
         }
      } // if if ( tid < NColumnOnce*PS2 )


#     else // CONSERVE_MASS


//    4b. full-step solution if CONSERVE_MASS is not defined (equivalent to the 3rd-order Taylor expansion)
      if ( tid < NColumnOnce*PS2 )
      {
         Re_New  = Re_Old - Coeff1*LAP1( s_In[1][ty], i ) - Coeff2*LAP2( s_In[0][ty], i ) + Coeff3*LAP3( s_In[1][ty], i );
         Im_New  = Im_Old + Coeff1*LAP1( s_In[0][ty], i ) - Coeff2*LAP2( s_In[1][ty], i ) - Coeff3*LAP3( s_In[0][ty], i );
         Amp_New = SQR( Re_New ) + SQR( Im_New );
      }


#     endif // CONSERVE_MASS ... else ...


//    5. store the updated data (and fluxes) back to the global memory
      if ( tid < NColumnOnce*PS2 )
      {
//       5.1 data
         if ( FinalOut )
         {
//          apply the the minimum density check
//          --> to be consistent with the CPU solver, we apply it just before storing the output results to g_Fluid_Out
            if ( Amp_New < MinDens )
            {
               const real Rescale = SQRT( MinDens / (real)Amp_New );

               Re_New *= Rescale;
               Im_New *= Rescale;
               Amp_New = MinDens;
            }

            switch ( XYZ )
            {
               case 0:  Idx2 = to1D2( k, j, i );    break;
               case 3:  Idx2 = to1D2( k, i, j );    break;
               case 6:  Idx2 = to1D2( i, k, j );    break;
            }

            g_Fluid_Out[bx][0][Idx2] = Amp_New;
            g_Fluid_Out[bx][1][Idx2] = Re_New;
            g_Fluid_Out[bx][2][Idx2] = Im_New;
         }

         else
         {
            g_Fluid_In[bx][0][Idx1] = Re_New;
            g_Fluid_In[bx][1][Idx1] = Im_New;
         }


//       5.2 fluxes (for the flux-correction operation)
         if ( StoreFlux  &&  tx == 0 )
         if ( k >= FLU_GHOST_SIZE  &&  k < FLU_NXT-FLU_GHOST_SIZE )
         if ( j >= FLU_GHOST_SIZE  &&  j < FLU_NXT-FLU_GHOST_SIZE )
         {
            Idx3 = __umul24( k-FLU_GHOST_SIZE, PS2 ) + (j-FLU_GHOST_SIZE);

            g_Flux[bx][XYZ+0][0][Idx3] = s_Flux[ty][  0]*_Eta2_dh;
            g_Flux[bx][XYZ+1][0][Idx3] = s_Flux[ty][PS1]*_Eta2_dh;
            g_Flux[bx][XYZ+2][0][Idx3] = s_Flux[ty][PS2]*_Eta2_dh;
         }


//       5.3 reset the target array indices
         j += NColumnOnce;

         if ( j >= j_end )
         {
            delta_k  = ( j - j_end )/size_j + 1;
            k       += delta_k;
            j       -= __umul24( size_j, delta_k );
         }
      } // if ( tid < NColumnOnce*PS2 )

      __syncthreads();

      Column0     += NColumnOnce;
      NColumnOnce  = MIN( NColumnTotal - Column0, FLU_BLOCK_SIZE_Y );

   } // while ( Column0 < NColumnTotal )

} // FUNCTION : CUFLU_Advance



#endif // #if ( defined GPU  &&  MODEL == ELBDM  &&  WAVE_SCHEME == WAVE_FD )
