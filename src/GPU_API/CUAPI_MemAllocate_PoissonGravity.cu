#include "CUAPI.h"

#if ( defined GPU  &&  defined GRAVITY )



extern real (*d_Rho_Array_P    )[ RHO_NXT*RHO_NXT*RHO_NXT ];
extern real (*d_Pot_Array_P_In )[ POT_NXT*POT_NXT*POT_NXT ];
extern real (*d_Pot_Array_P_Out)[ GRA_NXT*GRA_NXT*GRA_NXT ];
#ifdef UNSPLIT_GRAVITY
extern real (*d_Pot_Array_USG_G)[ USG_NXT_G*USG_NXT_G*USG_NXT_G ];
extern real (*d_Flu_Array_USG_G)[GRA_NIN_USG][ PS1*PS1*PS1 ];
#endif
extern real (*d_Flu_Array_G    )[GRA_NIN  ][ PS1*PS1*PS1 ];
extern double (*d_Corner_Array_G)[3];
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_G     )[ PS1*PS1*PS1 ];
#endif
extern real (*d_Pot_Array_T)    [ CUBE(GRA_NXT) ];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_PoissonGravity
// Description :  Allocate device and host memory for the Poisson and Gravity solvers
//
// Parameter   :  Pot_NPG  : Number of patch groups evaluated simultaneously by GPU
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_PoissonGravity( const int Pot_NPG )
{

   const long Pot_NP            = 8*Pot_NPG;
   const long Rho_MemSize_P     = sizeof(real  )*Pot_NP*CUBE(RHO_NXT);
   const long Pot_MemSize_P_In  = sizeof(real  )*Pot_NP*CUBE(POT_NXT);
   const long Pot_MemSize_P_Out = sizeof(real  )*Pot_NP*CUBE(GRA_NXT);
#  ifdef UNSPLIT_GRAVITY
   const long Pot_MemSize_USG_G = sizeof(real  )*Pot_NP*CUBE(USG_NXT_G);
   const long Flu_MemSize_USG_G = sizeof(real  )*Pot_NP*CUBE(PS1)*(GRA_NIN_USG);
#  endif
   const long Flu_MemSize_G     = sizeof(real  )*Pot_NP*CUBE(PS1)*(GRA_NIN  );
   const long Corner_MemSize    = sizeof(double)*Pot_NP*3;
#  ifdef DUAL_ENERGY
   const long DE_MemSize_G      = sizeof(char  )*Pot_NP*CUBE(PS1);
#  endif
   const long Pot_MemSize_T     = sizeof(real  )*Pot_NP*CUBE(GRA_NXT);


// output the total memory requirement
   long TotalSize = Rho_MemSize_P + Pot_MemSize_P_In + Pot_MemSize_P_Out + Flu_MemSize_G + Pot_MemSize_T;
#  ifdef UNSPLIT_GRAVITY
   TotalSize += Pot_MemSize_USG_G + Flu_MemSize_USG_G;
#  endif
#  ifdef DUAL_ENERGY
   TotalSize += DE_MemSize_G;
#  endif

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU Poisson and gravity solver = %ld MB\n",
                   TotalSize/(1<<20) );


// allocate the device memory
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Rho_Array_P,     Rho_MemSize_P     )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_P_In,  Pot_MemSize_P_In  )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_P_Out, Pot_MemSize_P_Out )  );
#  ifdef UNSPLIT_GRAVITY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_USG_G, Pot_MemSize_USG_G )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_USG_G, Flu_MemSize_USG_G )  );
#  endif
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_G,     Flu_MemSize_G     )  );

   if ( OPT__GRAVITY_TYPE == GRAVITY_EXTERNAL  ||  OPT__GRAVITY_TYPE == GRAVITY_BOTH  ||  OPT__EXTERNAL_POT )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Corner_Array_G,  Corner_MemSize    )  );

#  ifdef DUAL_ENERGY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_DE_Array_G,      DE_MemSize_G      )  );
#  endif

   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_T,     Pot_MemSize_T     )  );


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Rho_Array_P    [t], Rho_MemSize_P     )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_P_In [t], Pot_MemSize_P_In  )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_P_Out[t], Pot_MemSize_P_Out )  );
#     ifdef UNSPLIT_GRAVITY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_USG_G[t], Pot_MemSize_USG_G )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_USG_G[t], Flu_MemSize_USG_G )  );
#     endif
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_G    [t], Flu_MemSize_G     )  );

      if ( OPT__GRAVITY_TYPE == GRAVITY_EXTERNAL  ||  OPT__GRAVITY_TYPE == GRAVITY_BOTH  ||  OPT__EXTERNAL_POT )
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Corner_Array_G [t], Corner_MemSize    )  );

#     ifdef DUAL_ENERGY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_DE_Array_G     [t], DE_MemSize_G      )  );
#     endif

      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_T    [t], Pot_MemSize_T     )  );
   } // for (int t=0; t<2; t++)

} // FUNCTION : CUAPI_MemAllocate_PoissonGravity



#endif // #if ( defined GPU  &&  defined GRAVITY )
