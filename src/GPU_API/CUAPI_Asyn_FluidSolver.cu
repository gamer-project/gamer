#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU



#if   ( MODEL == HYDRO )
#if   ( FLU_SCHEME == RTVD )
__global__ void CUFLU_FluidSolver_RTVD(
   real g_Fluid_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
   real g_Fluid_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   real g_Flux     [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double g_Corner[][3],
   const real g_Pot_USG[][ CUBE(USG_NXT_F) ],
   const real dt, const real _dh, const bool StoreFlux,
   const bool XYZ, const real MinDens, const real MinPres, const real MinEint,
   const EoS_DE2P_t EoS_DensEint2Pres_Func,
   const EoS_DP2E_t EoS_DensPres2Eint_Func,
   const EoS_DP2C_t EoS_DensPres2CSqr_Func );
#elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
__global__
void CUFLU_FluidSolver_MHM(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch, const bool NormPassive, const int NNorm,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_DE2P_t EoS_DensEint2Pres_Func,
   const EoS_DP2E_t EoS_DensPres2Eint_Func,
   const EoS_DP2C_t EoS_DensPres2CSqr_Func,
   const EoS_GUESS_t EoS_GuessHTilde_Func,
   const EoS_H2TEM_t EoS_HTilde2Temp_Func,
   const EoS_TEM2H_t EoS_Temp2HTilde_Func,
   const EoS_TEM2C_t EoS_Temper2CSqr_Func );
#elif ( FLU_SCHEME == CTU )
__global__
void CUFLU_FluidSolver_CTU(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch, const bool NormPassive, const int NNorm,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_DE2P_t EoS_DensEint2Pres_Func,
   const EoS_DP2E_t EoS_DensPres2Eint_Func,
   const EoS_DP2C_t EoS_DensPres2CSqr_Func );
#endif // FLU_SCHEME

#elif ( MODEL == ELBDM )
__global__ void CUFLU_ELBDMSolver( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                   real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                   real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                                   const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                   const real Taylor3_Coeff, const bool XYZ, const real MinDens );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

#ifndef GRAVITY
static ExtAcc_t GPUExtAcc_Ptr = NULL;
#endif


// device pointers
extern real (*d_Flu_Array_F_In )[FLU_NIN ][ CUBE(FLU_NXT) ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ CUBE(PS2) ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ SQR(PS2) ];
extern double (*d_Corner_Array_F)[3];
#if ( MODEL == HYDRO )
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ CUBE(PS2) ];
#else
static char (*d_DE_Array_F_Out)[ CUBE(PS2) ] = NULL;
#endif
#ifdef MHD
extern real (*d_Mag_Array_F_In )[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_Mag_Array_F_Out)[NCOMP_MAG][ PS2P1*SQR(PS2)         ];
extern real (*d_Ele_Array      )[9][NCOMP_ELE][ PS2P1*PS2 ];
#else
static real (*d_Mag_Array_F_In )[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ] = NULL;
static real (*d_Mag_Array_F_Out)[NCOMP_MAG][ PS2P1*SQR(PS2)          ] = NULL;
static real (*d_Ele_Array      )[9][NCOMP_ELE][ PS2P1*PS2 ]            = NULL;
#endif
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)      [NCOMP_LR            ][ CUBE(FLU_NXT)     ];
extern real (*d_Slope_PPM)[3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ];
extern real (*d_FC_Var)   [6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR)    ];
extern real (*d_FC_Flux)  [3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX)   ];
#ifdef MHD
extern real (*d_FC_Mag_Half)[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_EC_Ele     )[NCOMP_MAG][ CUBE(N_EC_ELE)          ];
#else
static real (*d_FC_Mag_Half)[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ] = NULL;
static real (*d_EC_Ele     )[NCOMP_MAG][ CUBE(N_EC_ELE)          ] = NULL;
#endif // MHD
#endif // FLU_SCHEME
#endif // #if ( MODEL == HYDRO )

#ifdef UNSPLIT_GRAVITY
extern real (*d_Pot_Array_USG_F)[ CUBE(USG_NXT_F) ];
#else
static real (*d_Pot_Array_USG_F)[ CUBE(USG_NXT_F) ] = NULL;
#endif

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Asyn_FluidSolver
// Description :  1. MODEL == HYDRO : use GPU to solve the Euler equations by different schemes
//                                    --> invoke the kernel "CUFLU_FluidSolver_XXX"
//                2. MODEL == ELBDM : use GPU to solve the kinematic operator in the Schrodinger's equations
//                                    --> invoke the kernel "CUFLU_ELBDMSolver"
//
//                ***********************************************************
//                **                Asynchronous Function                  **
//                **                                                       **
//                **  will return before the execution in GPU is complete  **
//                ***********************************************************
//
// Note        :  1. Use streams for the asychronous memory copy between device and host
//                2. Prefix "d" : for pointers pointing to the "Device" memory space
//                   Prefix "h" : for pointers pointing to the "Host"   memory space
//                3. Use the input pamameter "XYZ" to control the order of update for dimensional-splitting
//                   method (currently only RTVD)
//                4. Currently five hydro schemes are supported :
//                   1. Relaxing TVD scheme                            (RTVD  ) -->   split
//                   2. MUSCL-Hancock scheme                           (MHM   ) --> unsplit
//                   3. MUSCL-Hancock scheme with Riemann prediction   (MHM_RP) --> unsplit
//                   4. Corner-Transport-Upwind scheme                 (CTU   ) --> unsplit
//
// Parameter   :  h_Flu_Array_In      : Host array to store the input fluid variables
//                h_Flu_Array_Out     : Host array to store the output fluid variables
//                h_Mag_Array_In      : Host array storing the input B field (for MHD only)
//                h_Mag_Array_Out     : Host array to store the output B field (for MHD only)
//                h_DE_Array_Out      : Host array to store the dual-energy status
//                h_Flux_Array        : Host array to store the output fluxes
//                h_Ele_Array         : Host array to store the output electric field (for MHD only)
//                h_Corner_Array      : Host array storing the physical corner coordinates of each patch group
//                h_Pot_Array_USG     : Host array storing the input potential for UNSPLIT_GRAVITY
//                NPatchGroup         : Number of patch groups evaluated simultaneously by GPU
//                dt                  : Time interval to advance solution
//                dh                  : Cell size
//                StoreFlux           : true --> store the coarse-fine fluxes
//                StoreElectric       : true --> store the coarse-fine electric field
//                XYZ                 : true  : x->y->z ( forward sweep)
//                                      false : z->y->x (backward sweep)
//                                      ~ useless in directionally unsplit schemes
//                LR_Limiter          : Slope limiter for the data reconstruction in the MHM/MHM_RP/CTU schemes
//                                      (0/1/2/3/4) = (vanLeer/generalized MinMod/vanAlbada/
//                                                     vanLeer + generalized MinMod/extrema-preserving) limiter
//                MinMod_Coeff        : Coefficient of the generalized MinMod limiter
//                ELBDM_Eta           : Particle mass / Planck constant
//                ELBDM_Taylor3_Coeff : Coefficient in front of the third term in the Taylor expansion for ELBDM
//                ELBDM_Taylor3_Auto  : true --> Determine ELBDM_Taylor3_Coeff automatically by invoking the
//                                               function "ELBDM_SetTaylor3Coeff"
//                Time                : Current physical time                      (for UNSPLIT_GRAVITY only)
//                UsePot              : Add self-gravity and/or external potential (for UNSPLIT_GRAVITY only)
//                ExtAcc              : Add external acceleration                  (for UNSPLIT_GRAVITY only)
//                MinDens/Pres/Eint   : Density, pressure, and internal energy floors
//                DualEnergySwitch    : Use the dual-energy formalism if E_int/E_kin < DualEnergySwitch
//                NormPassive         : true --> normalize passive scalars so that the sum of their mass density
//                                               is equal to the gas mass density
//                NNorm               : Number of passive scalars to be normalized
//                                      --> Should be set to the global variable "PassiveNorm_NVar"
//                JeansMinPres        : Apply minimum pressure estimated from the Jeans length
//                JeansMinPres_Coeff  : Coefficient used by JeansMinPres = G*(Jeans_NCell*Jeans_dh)^2/(Gamma*pi);
//                GPU_NStream         : Number of CUDA streams for the asynchronous memory copy
//-------------------------------------------------------------------------------------------------------
void CUAPI_Asyn_FluidSolver( real h_Flu_Array_In[][FLU_NIN ][ CUBE(FLU_NXT) ],
                             real h_Flu_Array_Out[][FLU_NOUT][ CUBE(PS2) ],
                             real h_Mag_Array_In[][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
                             real h_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
                             char h_DE_Array_Out[][ CUBE(PS2) ],
                             real h_Flux_Array[][9][NFLUX_TOTAL][ SQR(PS2) ],
                             real h_Ele_Array[][9][NCOMP_ELE][ PS2P1*PS2 ],
                             const double h_Corner_Array[][3],
                             real h_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
                             const int NPatchGroup, const real dt, const real dh,
                             const bool StoreFlux, const bool StoreElectric,
                             const bool XYZ, const LR_Limiter_t LR_Limiter, const real MinMod_Coeff,
                             const real ELBDM_Eta, real ELBDM_Taylor3_Coeff, const bool ELBDM_Taylor3_Auto,
                             const double Time, const bool UsePot, const OptExtAcc_t ExtAcc,
                             const real MinDens, const real MinPres, const real MinEint,
                             const real DualEnergySwitch, const bool NormPassive, const int NNorm,
                             const bool JeansMinPres, const real JeansMinPres_Coeff,
                             const int GPU_NStream )
{

// check
#  ifdef GAMER_DEBUG
#  if   ( MODEL == HYDRO )
   if ( LR_Limiter != VANLEER  &&  LR_Limiter != GMINMOD  &&  LR_Limiter != ALBADA  &&  LR_Limiter != EXTPRE  &&
        LR_Limiter != VL_GMINMOD  &&  LR_Limiter != LR_LIMITER_NONE )
      Aux_Error( ERROR_INFO, "unsupported limiter (%d) !!\n", LR_Limiter );

#  ifdef UNSPLIT_GRAVITY
   if ( UsePot )
   {
      if ( h_Pot_Array_USG   == NULL )   Aux_Error( ERROR_INFO, "h_Pot_Array_USG == NULL !!\n" );
      if ( d_Pot_Array_USG_F == NULL )   Aux_Error( ERROR_INFO, "d_Pot_Array_USG_F == NULL !!\n" );
   }

   if ( ExtAcc )
   {
      if ( h_Corner_Array   == NULL )    Aux_Error( ERROR_INFO, "h_Corner_Array == NULL !!\n" );
      if ( d_Corner_Array_F == NULL )    Aux_Error( ERROR_INFO, "d_Corner_Array_F == NULL !!\n" );
   }
#  endif

#  elif ( MODEL == ELBDM )

#  else
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif

   if ( StoreFlux )
   {
      if ( d_Flux_Array == NULL )   Aux_Error( ERROR_INFO, "d_Flux_Array == NULL !!\n" );
      if ( h_Flux_Array == NULL )   Aux_Error( ERROR_INFO, "h_Flux_Array == NULL !!\n" );
   }

#  ifdef MHD
   if ( h_Mag_Array_In    == NULL ) Aux_Error( ERROR_INFO, "h_Mag_Array_In == NULL !!\n" );
   if ( d_Mag_Array_F_In  == NULL ) Aux_Error( ERROR_INFO, "d_Mag_Array_F_In == NULL !!\n" );

   if ( h_Mag_Array_Out   == NULL ) Aux_Error( ERROR_INFO, "h_Mag_Array_Out == NULL !!\n" );
   if ( d_Mag_Array_F_Out == NULL ) Aux_Error( ERROR_INFO, "d_Mag_Array_F_Out == NULL !!\n" );

   if ( d_FC_Mag_Half     == NULL ) Aux_Error( ERROR_INFO, "d_FC_Mag_Half == NULL !!\n" );
   if ( d_EC_Ele          == NULL ) Aux_Error( ERROR_INFO, "d_EC_Ele == NULL !!\n" );

   if ( StoreElectric )
   {
      if ( d_Ele_Array == NULL )   Aux_Error( ERROR_INFO, "d_Ele_Array == NULL !!\n" );
      if ( h_Ele_Array == NULL )   Aux_Error( ERROR_INFO, "h_Ele_Array == NULL !!\n" );
   }
#  endif
#  endif // #ifdef GAMER_DEBUG


   const dim3 BlockDim_FluidSolver ( FLU_BLOCK_SIZE_X, FLU_BLOCK_SIZE_Y, 1 ); // for the fluidsolvers

// model-dependent operations
#  if   ( MODEL == HYDRO )

#  elif ( MODEL == ELBDM )
// evaluate the optimized Taylor expansion coefficient
   if ( ELBDM_Taylor3_Auto )  ELBDM_Taylor3_Coeff = ELBDM_SetTaylor3Coeff( dt, dh, ELBDM_Eta );

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL

   int *NPatch_per_Stream  = new int [GPU_NStream];
   int *UsedPatch          = new int [GPU_NStream];
   int *Flu_MemSize_In     = new int [GPU_NStream];
   int *Flu_MemSize_Out    = new int [GPU_NStream];
   int *Flux_MemSize       = new int [GPU_NStream];
#  ifdef MHD
   int *Mag_MemSize_In     = new int [GPU_NStream];
   int *Mag_MemSize_Out    = new int [GPU_NStream];
   int *Ele_MemSize        = new int [GPU_NStream];
#  endif
#  ifdef UNSPLIT_GRAVITY
   int *USG_MemSize        = new int [GPU_NStream];
   int *Corner_MemSize     = new int [GPU_NStream];
#  endif
#  ifdef DUAL_ENERGY
   int *DE_MemSize_Out     = new int [GPU_NStream];
#  endif


// set the number of patches of each stream
   UsedPatch[0] = 0;

   if ( GPU_NStream == 1 )    NPatch_per_Stream[0] = NPatchGroup;
   else
   {
      for (int s=0; s<GPU_NStream-1; s++)
      {
         NPatch_per_Stream[s] = NPatchGroup / GPU_NStream;
         UsedPatch[s+1] = UsedPatch[s] + NPatch_per_Stream[s];
      }

      NPatch_per_Stream[GPU_NStream-1] = NPatchGroup - UsedPatch[GPU_NStream-1];
   }


// set the size of data to be transferred into GPU in each stream
   for (int s=0; s<GPU_NStream; s++)
   {
      Flu_MemSize_In [s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NIN *CUBE(FLU_NXT);
      Flu_MemSize_Out[s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NOUT*CUBE(PS2);
      Flux_MemSize   [s] = sizeof(real  )*NPatch_per_Stream[s]*NFLUX_TOTAL*9*PS2*PS2;
#     ifdef MHD
      Mag_MemSize_In [s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_MAG*FLU_NXT_P1*SQR(FLU_NXT);
      Mag_MemSize_Out[s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_MAG*PS2P1*SQR(PS2);
      Ele_MemSize    [s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_ELE*9*PS2P1*PS2;
#     endif
#     ifdef UNSPLIT_GRAVITY
      USG_MemSize    [s] = sizeof(real  )*NPatch_per_Stream[s]*CUBE(USG_NXT_F);
      Corner_MemSize [s] = sizeof(double)*NPatch_per_Stream[s]*3;
#     endif
#     ifdef DUAL_ENERGY
      DE_MemSize_Out [s] = sizeof(char  )*NPatch_per_Stream[s]*CUBE(PS2);
#     endif
   }


// a. copy data from host to device
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Flu_Array_F_In  + UsedPatch[s], h_Flu_Array_In  + UsedPatch[s],
                         Flu_MemSize_In[s], hipMemcpyHostToDevice, Stream[s] )  );

#     ifdef MHD
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Mag_Array_F_In  + UsedPatch[s], h_Mag_Array_In  + UsedPatch[s],
                         Mag_MemSize_In[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif

#     ifdef UNSPLIT_GRAVITY
      if ( UsePot )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Pot_Array_USG_F + UsedPatch[s], h_Pot_Array_USG + UsedPatch[s],
                         USG_MemSize   [s], hipMemcpyHostToDevice, Stream[s] )  );

      if ( ExtAcc )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Corner_Array_F  + UsedPatch[s], h_Corner_Array  + UsedPatch[s],
                         Corner_MemSize[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif
   } // for (int s=0; s<GPU_NStream; s++)


// b. execute the kernel
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

#     if   ( MODEL == HYDRO )

#        if   ( FLU_SCHEME == RTVD )

         CUFLU_FluidSolver_RTVD <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              dt, 1.0/dh, StoreFlux, XYZ, MinDens, MinPres, MinEint,
              EoS_DensEint2Pres_GPUPtr, EoS_DensPres2Eint_GPUPtr, EoS_DensPres2CSqr_GPUPtr );

#        elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )

         CUFLU_FluidSolver_MHM <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Mag_Array_F_In  + UsedPatch[s],
              d_Mag_Array_F_Out + UsedPatch[s],
              d_DE_Array_F_Out  + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Ele_Array       + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              d_PriVar          + UsedPatch[s],
              d_Slope_PPM       + UsedPatch[s],
              d_FC_Var          + UsedPatch[s],
              d_FC_Flux         + UsedPatch[s],
              d_FC_Mag_Half     + UsedPatch[s],
              d_EC_Ele          + UsedPatch[s],
              dt, dh, StoreFlux, StoreElectric, LR_Limiter, MinMod_Coeff,
              Time, UsePot, ExtAcc, GPUExtAcc_Ptr, MinDens, MinPres, MinEint,
              DualEnergySwitch, NormPassive, NNorm, JeansMinPres, JeansMinPres_Coeff,
              EoS_DensEint2Pres_GPUPtr, EoS_DensPres2Eint_GPUPtr, EoS_DensPres2CSqr_GPUPtr,
              EoS_GuessHTilde_GPUPtr, EoS_HTilde2Temp_GPUPtr, EoS_Temp2HTilde_GPUPtr, EoS_Temper2CSqr_GPUPtr );

#        elif ( FLU_SCHEME == CTU )

         CUFLU_FluidSolver_CTU <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Mag_Array_F_In  + UsedPatch[s],
              d_Mag_Array_F_Out + UsedPatch[s],
              d_DE_Array_F_Out  + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Ele_Array       + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              d_PriVar          + UsedPatch[s],
              d_Slope_PPM       + UsedPatch[s],
              d_FC_Var          + UsedPatch[s],
              d_FC_Flux         + UsedPatch[s],
              d_FC_Mag_Half     + UsedPatch[s],
              d_EC_Ele          + UsedPatch[s],
              dt, dh, StoreFlux, StoreElectric, LR_Limiter, MinMod_Coeff,
              Time, UsePot, ExtAcc, GPUExtAcc_Ptr, MinDens, MinPres, MinEint,
              DualEnergySwitch, NormPassive, NNorm, JeansMinPres, JeansMinPres_Coeff,
              EoS_DensEint2Pres_GPUPtr, EoS_DensPres2Eint_GPUPtr, EoS_DensPres2CSqr_GPUPtr );

#        else

#        error : unsupported GPU hydro scheme

#        endif // FLU_SCHEME

#     elif ( MODEL == ELBDM )

         CUFLU_ELBDMSolver <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              dt, 1.0/dh, ELBDM_Eta, StoreFlux, ELBDM_Taylor3_Coeff, XYZ, MinDens );

#     else

#        error : unsupported MODEL !!

#     endif // MODEL


      CUDA_CHECK_ERROR( hipGetLastError() );
   } // for (int s=0; s<GPU_NStream; s++)


// c. copy data from device to host
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Flu_Array_Out + UsedPatch[s], d_Flu_Array_F_Out + UsedPatch[s],
                         Flu_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );

      if ( StoreFlux )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Flux_Array    + UsedPatch[s], d_Flux_Array      + UsedPatch[s],
                         Flux_MemSize[s],    hipMemcpyDeviceToHost, Stream[s] )  );

#     ifdef MHD
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Mag_Array_Out + UsedPatch[s], d_Mag_Array_F_Out + UsedPatch[s],
                         Mag_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );

      if ( StoreElectric )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Ele_Array     + UsedPatch[s], d_Ele_Array       + UsedPatch[s],
                         Ele_MemSize[s],    hipMemcpyDeviceToHost, Stream[s] )  );
#     endif

#     ifdef DUAL_ENERGY
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_DE_Array_Out  + UsedPatch[s], d_DE_Array_F_Out  + UsedPatch[s],
                         DE_MemSize_Out[s],  hipMemcpyDeviceToHost, Stream[s] )  );
#     endif
   } // for (int s=0; s<GPU_NStream; s++)


   delete [] NPatch_per_Stream;
   delete [] UsedPatch;
   delete [] Flu_MemSize_In;
   delete [] Flu_MemSize_Out;
   delete [] Flux_MemSize;
#  ifdef MHD
   delete [] Mag_MemSize_In;
   delete [] Mag_MemSize_Out;
   delete [] Ele_MemSize;
#  endif
#  ifdef UNSPLIT_GRAVITY
   delete [] USG_MemSize;
   delete [] Corner_MemSize;
#  endif
#  ifdef DUAL_ENERGY
   delete [] DE_MemSize_Out;
#  endif

} // FUNCTION : CUAPI_Asyn_FluidSolver



#endif // #ifdef GPU
