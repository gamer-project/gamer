#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU

#if   ( MODEL == HYDRO )
#if   ( FLU_SCHEME == RTVD )
__global__ void CUFLU_FluidSolver_RTVD(
   real g_Fluid_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
   real g_Fluid_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   real g_Flux     [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double g_Corner[][3],
   const real g_Pot_USG[][ CUBE(USG_NXT_F) ],
   const real dt, const real _dh, const bool StoreFlux,
   const bool XYZ, const real MinDens, const real MinPres, const real MinEint,
   const EoS_t EoS );
#elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
__global__
void CUFLU_FluidSolver_MHM(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const int MinMod_MaxIter, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool FracPassive, const int NFrac,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_t EoS, const MicroPhy_t MicroPhy );
#elif ( FLU_SCHEME == CTU )
__global__
void CUFLU_FluidSolver_CTU(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool FracPassive, const int NFrac,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_t EoS );
#endif // FLU_SCHEME

#elif ( MODEL == ELBDM )

#if   ( WAVE_SCHEME == WAVE_FD )
__global__ void CUFLU_ELBDMSolver_FD( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                                      real g_Fluid_Out[][FLU_NOUT][ CUBE(PS2) ],
                                      real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                      const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                      const real Taylor3_Coeff, const bool XYZ, const real MinDens );
real ELBDM_SetTaylor3Coeff( const real dt, const real dh, const real Eta );
#elif ( WAVE_SCHEME == WAVE_GRAMFE )
#if   ( GRAMFE_SCHEME == GRAMFE_FFT )
__launch_bounds__(FFT::max_threads_per_block)
__global__
void CUFLU_ELBDMSolver_GramFE_FFT( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                                   real g_Fluid_Out[][FLU_NOUT ][ CUBE(PS2) ],
                                   real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                   const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                   const bool XYZ, const real MinDens,
                                   typename FFT::workspace_type workspace,
                                   typename IFFT::workspace_type workspace_inverse );
#elif ( GRAMFE_SCHEME == GRAMFE_MATMUL )
void ELBDM_GramFE_ComputeTimeEvolutionMatrix( gramfe_matmul_float (*output)[ 2*FLU_NXT ], const real dt, const real dh, const real Eta );
__global__
void CUFLU_ELBDMSolver_GramFE_MATMUL( real g_Fluid_In [][FLU_NIN ][ CUBE(FLU_NXT) ],
                                      real g_Fluid_Out[][FLU_NOUT ][ CUBE(PS2) ],
                                      real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                      gramfe_matmul_float g_Evolve[][ FLU_NXT*2 ],
                                      const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                      const bool XYZ, const real MinDens );
#else
#  error : ERROR : unsupported GRAMFE_SCHEME !!
#endif // GRAMFE_SCHEME
#else
#  error : ERROR : unsupported WAVE_SCHEME !!
#endif // WAVE_SCHEME

#if ( ELBDM_SCHEME == ELBDM_HYBRID )
__global__ void CUFLU_ELBDMSolver_HamiltonJacobi( real g_Fluid_In [][FLU_NIN ][ CUBE(HYB_NXT) ],
#                                                 ifdef GAMER_DEBUG
                                                  real g_Fluid_Out[][FLU_NOUT ][ CUBE(PS2) ],
#                                                 else
                                                  real g_Fluid_Out[][FLU_NIN ][ CUBE(PS2) ],
#                                                 endif
                                                  real g_Flux     [][9][NFLUX_TOTAL][ SQR(PS2) ],
                                                  const bool h_IsCompletelyRefined[],
                                                  const bool h_HasWaveCounterpart[][ CUBE(HYB_NXT) ],
                                                  const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                                  const bool XYZ, const real MinDens );
#endif

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

#if ( !defined GRAVITY  &&  MODEL == HYDRO )
static ExtAcc_t GPUExtAcc_Ptr = NULL;
#endif


// device pointers
extern real (*d_Flu_Array_F_In )[FLU_NIN ][ CUBE(FLU_NXT) ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ CUBE(PS2) ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ SQR(PS2) ];
extern double (*d_Corner_Array_F)[3];
#if ( MODEL == HYDRO )
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ CUBE(PS2) ];
#else
static char (*d_DE_Array_F_Out)[ CUBE(PS2) ] = NULL;
#endif
#ifdef MHD
extern real (*d_Mag_Array_F_In )[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_Mag_Array_F_Out)[NCOMP_MAG][ PS2P1*SQR(PS2)         ];
extern real (*d_Ele_Array      )[9][NCOMP_ELE][ PS2P1*PS2 ];
#else
static real (*d_Mag_Array_F_In )[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ] = NULL;
static real (*d_Mag_Array_F_Out)[NCOMP_MAG][ PS2P1*SQR(PS2)          ] = NULL;
static real (*d_Ele_Array      )[9][NCOMP_ELE][ PS2P1*PS2 ]            = NULL;
#endif
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)      [NCOMP_LR            ][ CUBE(FLU_NXT)     ];
extern real (*d_Slope_PPM)[3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ];
extern real (*d_FC_Var)   [6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR)    ];
extern real (*d_FC_Flux)  [3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX)   ];
#ifdef MHD
extern real (*d_FC_Mag_Half)[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_EC_Ele     )[NCOMP_MAG][ CUBE(N_EC_ELE)          ];
#else
static real (*d_FC_Mag_Half)[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ] = NULL;
static real (*d_EC_Ele     )[NCOMP_MAG][ CUBE(N_EC_ELE)          ] = NULL;
#endif // MHD
#endif // FLU_SCHEME
#endif // #if ( MODEL == HYDRO )

#if ( MODEL == ELBDM )
extern bool (*d_IsCompletelyRefined);
#endif

#if ( ELBDM_SCHEME == ELBDM_HYBRID )
extern bool (*d_HasWaveCounterpart)[ CUBE(HYB_NXT) ];
#endif // #if ( ELBDM_SCHEME == ELBDM_HYBRID )

#if ( GRAMFE_SCHEME == GRAMFE_MATMUL )
extern gramfe_matmul_float (*d_Flu_TimeEvo)[2 * FLU_NXT];
#endif // #if ( GRAMFE_SCHEME == GRAMFE_MATMUL )

#ifdef UNSPLIT_GRAVITY
extern real (*d_Pot_Array_USG_F)[ CUBE(USG_NXT_F) ];
#elif ( MODEL == HYDRO )
static real (*d_Pot_Array_USG_F)[ CUBE(USG_NXT_F) ] = NULL;
#endif

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Asyn_FluidSolver
// Description :  1. MODEL == HYDRO : use GPU to solve the Euler equations by different schemes
//                                    --> invoke the kernel "CUFLU_FluidSolver_XXX"
//                2. MODEL == ELBDM : use GPU to solve the kinematic operator in the Schrodinger's equations
//                                    --> invoke the kernel "CUFLU_ELBDMSolver_XXX"
//
//                ***********************************************************
//                **                Asynchronous Function                  **
//                **                                                       **
//                **  will return before the execution in GPU is complete  **
//                ***********************************************************
//
// Note        :  1. Use streams for the asychronous memory copy between device and host
//                2. Prefix "d" : for pointers pointing to the "Device" memory space
//                   Prefix "h" : for pointers pointing to the "Host"   memory space
//                3. Use the input pamameter "XYZ" to control the order of update for dimensional-splitting
//                   method (currently only RTVD)
//                4. Currently five hydro schemes are supported :
//                   1. Relaxing TVD scheme                            (RTVD  ) -->   split
//                   2. MUSCL-Hancock scheme                           (MHM   ) --> unsplit
//                   3. MUSCL-Hancock scheme with Riemann prediction   (MHM_RP) --> unsplit
//                   4. Corner-Transport-Upwind scheme                 (CTU   ) --> unsplit
//
// Parameter   :  h_Flu_Array_In        : Host array to store the input fluid variables
//                h_Flu_Array_Out       : Host array to store the output fluid variables
//                h_Mag_Array_In        : Host array storing the input B field (for MHD only)
//                h_Mag_Array_Out       : Host array to store the output B field (for MHD only)
//                h_DE_Array_Out        : Host array to store the dual-energy status
//                h_Flux_Array          : Host array to store the output fluxes
//                h_Ele_Array           : Host array to store the output electric field (for MHD only)
//                h_Corner_Array        : Host array storing the physical corner coordinates of each patch group
//                h_Pot_Array_USG       : Host array storing the input potential for UNSPLIT_GRAVITY
//                h_IsCompletelyRefined : Host array storing which patch groups are completely refined ( ELBDM only )
//                h_HasWaveCounterpart  : Host array storing which cells have wave counterpart ( ELBDM_HYBRID only )
//                NPatchGroup           : Number of patch groups evaluated simultaneously by GPU
//                dt                    : Time interval to advance solution
//                dh                    : Cell size
//                StoreFlux             : true --> store the coarse-fine fluxes
//                StoreElectric         : true --> store the coarse-fine electric field
//                XYZ                   : true  : x->y->z ( forward sweep)
//                                        false : z->y->x (backward sweep)
//                                        ~ useless in directionally unsplit schemes
//                LR_Limiter            : Slope limiter for the data reconstruction in the MHM/MHM_RP/CTU schemes
//                                        (0/1/2/3/4) = (vanLeer/generalized MinMod/vanAlbada/
//                                                       vanLeer + generalized MinMod/extrema-preserving) limiter
//                MinMod_Coeff          : Coefficient of the generalized MinMod limiter
//                MinMod_MaxIter        : Maximum number of iterations to reduce MinMod_Coeff
//                ELBDM_Eta             : Particle mass / Planck constant
//                ELBDM_Taylor3_Coeff   : Coefficient in front of the third term in the Taylor expansion for ELBDM
//                ELBDM_Taylor3_Auto    : true --> Determine ELBDM_Taylor3_Coeff automatically by invoking the
//                                                 function "ELBDM_SetTaylor3Coeff"
//                Time                  : Current physical time                      (for UNSPLIT_GRAVITY only)
//                UsePot                : Add self-gravity and/or external potential (for UNSPLIT_GRAVITY only)
//                ExtAcc                : Add external acceleration                  (for UNSPLIT_GRAVITY only)
//                MicroPhy              : Microphysics object
//                MinDens/Pres/Eint     : Density, pressure, and internal energy floors
//                DualEnergySwitch      : Use the dual-energy formalism if E_int/E_kin < DualEnergySwitch
//                NormPassive           : true --> normalize passive scalars so that the sum of their mass density
//                                                 is equal to the gas mass density
//                NNorm                 : Number of passive scalars to be normalized
//                                        --> Should be set to the global variable "PassiveNorm_NVar"
//                FracPassive           : true --> convert passive scalars to mass fraction during data reconstruction
//                NFrac                 : Number of passive scalars for the option "FracPassive"
//                                        --> Should be set to the global variable "PassiveIntFrac_NVar"
//                JeansMinPres          : Apply minimum pressure estimated from the Jeans length
//                JeansMinPres_Coeff    : Coefficient used by JeansMinPres = G*(Jeans_NCell*Jeans_dh)^2/(Gamma*pi);
//                GPU_NStream           : Number of CUDA streams for the asynchronous memory copy
//                UseWaveFlag           : Determine whether to use wave or phase scheme
//-------------------------------------------------------------------------------------------------------
void CUAPI_Asyn_FluidSolver( real h_Flu_Array_In[][FLU_NIN ][ CUBE(FLU_NXT) ],
                             real h_Flu_Array_Out[][FLU_NOUT][ CUBE(PS2) ],
                             real h_Mag_Array_In[][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
                             real h_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
                             char h_DE_Array_Out[][ CUBE(PS2) ],
                             real h_Flux_Array[][9][NFLUX_TOTAL][ SQR(PS2) ],
                             real h_Ele_Array[][9][NCOMP_ELE][ PS2P1*PS2 ],
                             const double h_Corner_Array[][3],
                             real h_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
                             const bool h_IsCompletelyRefined[],
                             const bool h_HasWaveCounterpart[][ CUBE(HYB_NXT) ],
                             const int NPatchGroup, const real dt, const real dh,
                             const bool StoreFlux, const bool StoreElectric,
                             const bool XYZ, const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const int MinMod_MaxIter,
                             const real ELBDM_Eta, real ELBDM_Taylor3_Coeff, const bool ELBDM_Taylor3_Auto,
                             const double Time, const bool UsePot, const OptExtAcc_t ExtAcc, const MicroPhy_t MicroPhy,
                             const real MinDens, const real MinPres, const real MinEint,
                             const real DualEnergySwitch,
                             const bool NormPassive, const int NNorm,
                             const bool FracPassive, const int NFrac,
                             const bool JeansMinPres, const real JeansMinPres_Coeff,
                             const int GPU_NStream, const bool UseWaveFlag )
{

// check
#  ifdef GAMER_DEBUG
#  if   ( MODEL == HYDRO )

#  ifdef UNSPLIT_GRAVITY
   if ( UsePot )
   {
      if ( h_Pot_Array_USG   == NULL )    Aux_Error( ERROR_INFO, "h_Pot_Array_USG == NULL !!\n" );
      if ( d_Pot_Array_USG_F == NULL )    Aux_Error( ERROR_INFO, "d_Pot_Array_USG_F == NULL !!\n" );
   }

   if ( ExtAcc )
   {
      if ( h_Corner_Array   == NULL )     Aux_Error( ERROR_INFO, "h_Corner_Array == NULL !!\n" );
      if ( d_Corner_Array_F == NULL )     Aux_Error( ERROR_INFO, "d_Corner_Array_F == NULL !!\n" );
   }
#  endif

#  elif ( MODEL == ELBDM )
   if ( h_IsCompletelyRefined == NULL )   Aux_Error( ERROR_INFO, "h_IsCompletelyRefined == NULL !!\n" );

#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   if ( h_HasWaveCounterpart == NULL  &&  !UseWaveFlag )
                                          Aux_Error( ERROR_INFO, "h_HasWaveCounterpart == NULL !!\n" );
#  endif

#  else
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif

   if ( StoreFlux )
   {
      if ( d_Flux_Array == NULL )   Aux_Error( ERROR_INFO, "d_Flux_Array == NULL !!\n" );
      if ( h_Flux_Array == NULL )   Aux_Error( ERROR_INFO, "h_Flux_Array == NULL !!\n" );
   }

#  ifdef MHD
   if ( h_Mag_Array_In    == NULL ) Aux_Error( ERROR_INFO, "h_Mag_Array_In == NULL !!\n" );
   if ( d_Mag_Array_F_In  == NULL ) Aux_Error( ERROR_INFO, "d_Mag_Array_F_In == NULL !!\n" );

   if ( h_Mag_Array_Out   == NULL ) Aux_Error( ERROR_INFO, "h_Mag_Array_Out == NULL !!\n" );
   if ( d_Mag_Array_F_Out == NULL ) Aux_Error( ERROR_INFO, "d_Mag_Array_F_Out == NULL !!\n" );

   if ( d_FC_Mag_Half     == NULL ) Aux_Error( ERROR_INFO, "d_FC_Mag_Half == NULL !!\n" );
   if ( d_EC_Ele          == NULL ) Aux_Error( ERROR_INFO, "d_EC_Ele == NULL !!\n" );

   if ( StoreElectric )
   {
      if ( d_Ele_Array == NULL )   Aux_Error( ERROR_INFO, "d_Ele_Array == NULL !!\n" );
      if ( h_Ele_Array == NULL )   Aux_Error( ERROR_INFO, "h_Ele_Array == NULL !!\n" );
   }
#  endif
#  endif // #ifdef GAMER_DEBUG


// thread block size
#  if (  !( MODEL == ELBDM  &&  WAVE_SCHEME == WAVE_GRAMFE  &&  GRAMFE_SCHEME == GRAMFE_FFT )  )
   const dim3 BlockDim_FluidSolver    ( FLU_BLOCK_SIZE_X, FLU_BLOCK_SIZE_Y,    1 ); // for the fluid solvers
#  endif
#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   const dim3 BlockDim_FluidSolver_HJ ( FLU_BLOCK_SIZE_X, FLU_HJ_BLOCK_SIZE_Y, 1 ); // for the HJ solver
#  endif


// model-dependent operations
#  if   ( MODEL == HYDRO )

#  elif ( MODEL == ELBDM )

#  if ( WAVE_SCHEME == WAVE_GRAMFE  &&  GRAMFE_SCHEME == GRAMFE_FFT )
   uint cufftdx_shared_memory_size = NULL_INT;
#  endif

#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   if ( UseWaveFlag ) {
#  endif

#  if   ( WAVE_SCHEME == WAVE_FD )

// evaluate the optimized Taylor expansion coefficient
   if ( ELBDM_Taylor3_Auto )  ELBDM_Taylor3_Coeff = ELBDM_SetTaylor3Coeff( dt, dh, ELBDM_Eta );

#  elif ( WAVE_SCHEME == WAVE_GRAMFE )

// set up GPU FFT if GPU is used for Gram Fourier extension FFT scheme
#  if   ( GRAMFE_SCHEME == GRAMFE_FFT )
// total size of shared memory required for storing FFT::ffts_per_block rows of data after Gram extension and the coefficients of the respective left and right extension polynomials
   auto size       = FFT::ffts_per_block*cufftdx::size_of<FFT>::value + 2*FFT::ffts_per_block*GRAMFE_NDELTA;
   auto size_bytes = size*sizeof(complex_type);

// shared memory must fit input data and must be big enough to run FFT
   cufftdx_shared_memory_size = std::max( (unsigned int)FFT::shared_memory_size, (unsigned int)size_bytes );

// increase max shared memory if needed
   CUDA_CHECK_ERROR(  hipFuncSetAttribute(reinterpret_cast<const void*>( CUFLU_ELBDMSolver_GramFE_FFT), hipFuncAttributeMaxDynamicSharedMemorySize,
                                            cufftdx_shared_memory_size )  );

#  elif ( GRAMFE_SCHEME == GRAMFE_MATMUL )
// time evolution matrix is copied to GPU in InvokeSolver()

#  else
#     error : ERROR : unsupported GRAMFE_SCHEME !!
#  endif // GRAMFE_SCHEME

#  else
#     error : ERROR : unsupported WAVE_SCHEME !!
#  endif // WAVE_SCHEME

#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   } // if ( UseWaveFlag )
#  endif

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL

   int *NPatch_per_Stream               = new int [GPU_NStream];
   int *UsedPatch                       = new int [GPU_NStream];
   int *Flu_MemSize_In                  = new int [GPU_NStream];
   int *Flu_MemSize_Out                 = new int [GPU_NStream];
   int *Flux_MemSize                    = new int [GPU_NStream];
#  ifdef MHD
   int *Mag_MemSize_In                  = new int [GPU_NStream];
   int *Mag_MemSize_Out                 = new int [GPU_NStream];
   int *Ele_MemSize                     = new int [GPU_NStream];
#  endif
#  ifdef UNSPLIT_GRAVITY
   int *USG_MemSize                     = new int [GPU_NStream];
   int *Corner_MemSize                  = new int [GPU_NStream];
#  endif
#  ifdef DUAL_ENERGY
   int *DE_MemSize_Out                  = new int [GPU_NStream];
#  endif
#  if ( MODEL == ELBDM )
   int *Flu_MemSize_IsCompletelyRefined = new int [GPU_NStream];
#  endif
#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   int *Flu_MemSize_HasWaveCounterpart  = ( !UseWaveFlag ) ? new int [GPU_NStream] : NULL;
#  endif


// set the number of patches of each stream
   UsedPatch[0] = 0;

   if ( GPU_NStream == 1 )    NPatch_per_Stream[0] = NPatchGroup;
   else
   {
      for (int s=0; s<GPU_NStream-1; s++)
      {
         NPatch_per_Stream[s] = NPatchGroup / GPU_NStream;
         UsedPatch[s+1] = UsedPatch[s] + NPatch_per_Stream[s];
      }

      NPatch_per_Stream[GPU_NStream-1] = NPatchGroup - UsedPatch[GPU_NStream-1];
   }


// set the size of data to be transferred into GPU in each stream
   for (int s=0; s<GPU_NStream; s++)
   {
      Flu_MemSize_In [s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NIN *CUBE(FLU_NXT);
      Flu_MemSize_Out[s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NOUT*CUBE(PS2);
      Flux_MemSize   [s] = sizeof(real  )*NPatch_per_Stream[s]*NFLUX_TOTAL*9*SQR(PS2);
#     ifdef MHD
      Mag_MemSize_In [s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_MAG*FLU_NXT_P1*SQR(FLU_NXT);
      Mag_MemSize_Out[s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_MAG*PS2P1*SQR(PS2);
      Ele_MemSize    [s] = sizeof(real  )*NPatch_per_Stream[s]*NCOMP_ELE*9*PS2P1*PS2;
#     endif
#     ifdef UNSPLIT_GRAVITY
      USG_MemSize    [s] = sizeof(real  )*NPatch_per_Stream[s]*CUBE(USG_NXT_F);
      Corner_MemSize [s] = sizeof(double)*NPatch_per_Stream[s]*3;
#     endif
#     ifdef DUAL_ENERGY
      DE_MemSize_Out [s] = sizeof(char  )*NPatch_per_Stream[s]*CUBE(PS2);
#     endif

//    optimization for phase scheme:
//    (a) transfer CUBE(HYB_NXT) instead of CUBE(FLU_NXT) cells to GPU
//    (b) when not in the debug mode, do not transfer STUB back from GPU (so only FLU_NIN instead of FLU_NOUT components)
#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      if ( !UseWaveFlag ) {
      Flu_MemSize_In [s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NIN*CUBE(HYB_NXT);
#     ifndef GAMER_DEBUG
      Flu_MemSize_Out[s] = sizeof(real  )*NPatch_per_Stream[s]*FLU_NIN*CUBE(PS2);
#     endif
      }
#     endif // #if ( ELBDM_SCHEME == ELBDM_HYBRID )

#     if ( MODEL == ELBDM )
      Flu_MemSize_IsCompletelyRefined[s] = sizeof(bool)*NPatch_per_Stream[s];
#     endif
#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      if ( !UseWaveFlag )
      Flu_MemSize_HasWaveCounterpart [s] = sizeof(bool)*NPatch_per_Stream[s]*CUBE(HYB_NXT);
#     endif
   } // for (int s=0; s<GPU_NStream; s++)


// a. copy data from host to device
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      if ( UseWaveFlag ) {
#     endif
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Flu_Array_F_In  + UsedPatch[s], h_Flu_Array_In  + UsedPatch[s],
                         Flu_MemSize_In[s], hipMemcpyHostToDevice, Stream[s] )  );
#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      } else {
      real (*smaller_d_Flu_Array_F_In)[FLU_NIN][CUBE(HYB_NXT)] = (real (*)[FLU_NIN][CUBE(HYB_NXT)]) d_Flu_Array_F_In;
      real (*smaller_h_Flu_Array_In  )[FLU_NIN][CUBE(HYB_NXT)] = (real (*)[FLU_NIN][CUBE(HYB_NXT)]) h_Flu_Array_In  ;

      CUDA_CHECK_ERROR(  hipMemcpyAsync( smaller_d_Flu_Array_F_In + UsedPatch[s], smaller_h_Flu_Array_In + UsedPatch[s],
                         Flu_MemSize_In[s], hipMemcpyHostToDevice, Stream[s] )  );
      }
#     endif
#     ifdef MHD
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Mag_Array_F_In  + UsedPatch[s], h_Mag_Array_In  + UsedPatch[s],
                         Mag_MemSize_In[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif

#     ifdef UNSPLIT_GRAVITY
      if ( UsePot )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Pot_Array_USG_F + UsedPatch[s], h_Pot_Array_USG + UsedPatch[s],
                         USG_MemSize   [s], hipMemcpyHostToDevice, Stream[s] )  );

      if ( ExtAcc )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Corner_Array_F  + UsedPatch[s], h_Corner_Array  + UsedPatch[s],
                         Corner_MemSize[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif


#     if ( MODEL == ELBDM )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_IsCompletelyRefined + UsedPatch[s], h_IsCompletelyRefined + UsedPatch[s],
                         Flu_MemSize_IsCompletelyRefined[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif
#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      if ( !UseWaveFlag )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( d_HasWaveCounterpart  + UsedPatch[s], h_HasWaveCounterpart  + UsedPatch[s],
                         Flu_MemSize_HasWaveCounterpart[s], hipMemcpyHostToDevice, Stream[s] )  );
#     endif
   } // for (int s=0; s<GPU_NStream; s++)


// b. execute the kernel
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

#     if   ( MODEL == HYDRO )

#        if   ( FLU_SCHEME == RTVD )

         CUFLU_FluidSolver_RTVD <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              dt, 1.0/dh, StoreFlux, XYZ, MinDens, MinPres, MinEint, EoS );

#        elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )

         CUFLU_FluidSolver_MHM <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Mag_Array_F_In  + UsedPatch[s],
              d_Mag_Array_F_Out + UsedPatch[s],
              d_DE_Array_F_Out  + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Ele_Array       + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              d_PriVar          + UsedPatch[s],
              d_Slope_PPM       + UsedPatch[s],
              d_FC_Var          + UsedPatch[s],
              d_FC_Flux         + UsedPatch[s],
              d_FC_Mag_Half     + UsedPatch[s],
              d_EC_Ele          + UsedPatch[s],
              dt, dh, StoreFlux, StoreElectric, LR_Limiter, MinMod_Coeff, MinMod_MaxIter,
              Time, UsePot, ExtAcc, GPUExtAcc_Ptr, MinDens, MinPres, MinEint,
              DualEnergySwitch, NormPassive, NNorm, FracPassive, NFrac,
              JeansMinPres, JeansMinPres_Coeff, EoS, MicroPhy );

#        elif ( FLU_SCHEME == CTU )

         CUFLU_FluidSolver_CTU <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Mag_Array_F_In  + UsedPatch[s],
              d_Mag_Array_F_Out + UsedPatch[s],
              d_DE_Array_F_Out  + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Ele_Array       + UsedPatch[s],
              d_Corner_Array_F  + UsedPatch[s],
              d_Pot_Array_USG_F + UsedPatch[s],
              d_PriVar          + UsedPatch[s],
              d_Slope_PPM       + UsedPatch[s],
              d_FC_Var          + UsedPatch[s],
              d_FC_Flux         + UsedPatch[s],
              d_FC_Mag_Half     + UsedPatch[s],
              d_EC_Ele          + UsedPatch[s],
              dt, dh, StoreFlux, StoreElectric, LR_Limiter, MinMod_Coeff,
              Time, UsePot, ExtAcc, GPUExtAcc_Ptr, MinDens, MinPres, MinEint,
              DualEnergySwitch, NormPassive, NNorm, FracPassive, NFrac,
              JeansMinPres, JeansMinPres_Coeff, EoS );

#        else

#        error : unsupported GPU hydro scheme

#        endif // FLU_SCHEME

#     elif ( MODEL == ELBDM )

#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      if ( UseWaveFlag ) {
#     endif

#     if   ( WAVE_SCHEME == WAVE_FD )

         CUFLU_ELBDMSolver_FD <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              dt, 1.0/dh, ELBDM_Eta, StoreFlux, ELBDM_Taylor3_Coeff, XYZ, MinDens );

#     elif ( WAVE_SCHEME == WAVE_GRAMFE )

#     if ( GRAMFE_SCHEME == GRAMFE_FFT )

//       create forward and backward cufftx workspaces
         hipError_t error_code  = hipSuccess;
         FFT::workspace_type cufftdx_workspace  = cufftdx::make_workspace<FFT>( error_code );
         CUDA_CHECK_ERROR(error_code);
         error_code              = hipSuccess;
         IFFT::workspace_type cufftdx_iworkspace = cufftdx::make_workspace<IFFT>( error_code );
         CUDA_CHECK_ERROR(error_code);

         CUFLU_ELBDMSolver_GramFE_FFT <<< NPatch_per_Stream[s], FFT::block_dim, cufftdx_shared_memory_size, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              dt, 1.0/dh, ELBDM_Eta, StoreFlux, XYZ, MinDens, cufftdx_workspace, cufftdx_iworkspace );

#     elif ( GRAMFE_SCHEME == GRAMFE_MATMUL )
         CUFLU_ELBDMSolver_GramFE_MATMUL <<< NPatch_per_Stream[s], BlockDim_FluidSolver, 0, Stream[s] >>>
            ( d_Flu_Array_F_In  + UsedPatch[s],
              d_Flu_Array_F_Out + UsedPatch[s],
              d_Flux_Array      + UsedPatch[s],
              d_Flu_TimeEvo,
              dt, dh, ELBDM_Eta, StoreFlux, XYZ, MinDens );
#     else
#        error : ERROR : unsupported GRAMFE_SCHEME !!
#     endif // GRAMFE_SCHEME

#     else // WAVE_SCHEME
#        error : ERROR : unsupported WAVE_SCHEME !!
#     endif // WAVE_SCHEME

#     if ( ELBDM_SCHEME == ELBDM_HYBRID )
      } else {
         real (*smaller_d_Flu_Array_F_In) [FLU_NIN ][CUBE(HYB_NXT)] = (real (*)[FLU_NIN][CUBE(HYB_NXT)]) d_Flu_Array_F_In;
#        ifdef GAMER_DEBUG
         real (*smaller_d_Flu_Array_F_Out)[FLU_NOUT][CUBE(PS2)]     = d_Flu_Array_F_Out;
#        else
         real (*smaller_d_Flu_Array_F_Out)[FLU_NIN ][CUBE(PS2)]     = (real (*)[FLU_NIN][CUBE(PS2)]    ) d_Flu_Array_F_Out;
#        endif

         CUFLU_ELBDMSolver_HamiltonJacobi <<< NPatch_per_Stream[s], BlockDim_FluidSolver_HJ, 0, Stream[s] >>>
            (  smaller_d_Flu_Array_F_In  + UsedPatch[s],
               smaller_d_Flu_Array_F_Out + UsedPatch[s],
               d_Flux_Array              + UsedPatch[s],
               d_IsCompletelyRefined     + UsedPatch[s],
               d_HasWaveCounterpart      + UsedPatch[s],
               dt, 1.0/dh, ELBDM_Eta, StoreFlux, XYZ, MinDens );

      } // if ( UseWaveFlag ) ... else ...
#     endif // #if ( ELBDM_SCHEME == ELBDM_HYBRID )

#     else

#        error : unsupported MODEL !!

#     endif // MODEL


      CUDA_CHECK_ERROR( hipGetLastError() );
   } // for (int s=0; s<GPU_NStream; s++)


// c. copy data from device to host
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

#     if ( ELBDM_SCHEME == ELBDM_HYBRID  &&  !defined(GAMER_DEBUG) )
      if ( UseWaveFlag ) {
#     endif
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Flu_Array_Out + UsedPatch[s], d_Flu_Array_F_Out + UsedPatch[s],
                         Flu_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );
#     if ( ELBDM_SCHEME == ELBDM_HYBRID  &&  !defined(GAMER_DEBUG) )
      } else {
      real (*smaller_h_Flu_Array_Out  )[FLU_NIN][CUBE(PS2)] = (real (*)[FLU_NIN][CUBE(PS2)]) h_Flu_Array_Out;
      real (*smaller_d_Flu_Array_F_Out)[FLU_NIN][CUBE(PS2)] = (real (*)[FLU_NIN][CUBE(PS2)]) d_Flu_Array_F_Out;
      CUDA_CHECK_ERROR(  hipMemcpyAsync( smaller_h_Flu_Array_Out + UsedPatch[s], smaller_d_Flu_Array_F_Out + UsedPatch[s],
                         Flu_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );
      }
#     endif

      if ( StoreFlux )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Flux_Array    + UsedPatch[s], d_Flux_Array      + UsedPatch[s],
                         Flux_MemSize[s],    hipMemcpyDeviceToHost, Stream[s] )  );

#     ifdef MHD
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Mag_Array_Out + UsedPatch[s], d_Mag_Array_F_Out + UsedPatch[s],
                         Mag_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );

      if ( StoreElectric )
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Ele_Array     + UsedPatch[s], d_Ele_Array       + UsedPatch[s],
                         Ele_MemSize[s],    hipMemcpyDeviceToHost, Stream[s] )  );
#     endif

#     ifdef DUAL_ENERGY
      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_DE_Array_Out  + UsedPatch[s], d_DE_Array_F_Out  + UsedPatch[s],
                         DE_MemSize_Out[s],  hipMemcpyDeviceToHost, Stream[s] )  );
#     endif
   } // for (int s=0; s<GPU_NStream; s++)


   delete [] NPatch_per_Stream;
   delete [] UsedPatch;
   delete [] Flu_MemSize_In;
   delete [] Flu_MemSize_Out;
   delete [] Flux_MemSize;
#  ifdef MHD
   delete [] Mag_MemSize_In;
   delete [] Mag_MemSize_Out;
   delete [] Ele_MemSize;
#  endif
#  ifdef UNSPLIT_GRAVITY
   delete [] USG_MemSize;
   delete [] Corner_MemSize;
#  endif
#  ifdef DUAL_ENERGY
   delete [] DE_MemSize_Out;
#  endif
#  if ( MODEL == ELBDM )
   delete [] Flu_MemSize_IsCompletelyRefined;
#  endif
#  if ( ELBDM_SCHEME == ELBDM_HYBRID )
   delete [] Flu_MemSize_HasWaveCounterpart;
#  endif

} // FUNCTION : CUAPI_Asyn_FluidSolver



#endif // #ifdef GPU
