#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "CUFLU.h"
#ifdef GRAVITY
#include "CUPOT.h"
#endif

#ifdef GPU



// fluid solver prototypes in different models
#if   ( MODEL == HYDRO )
#if   ( FLU_SCHEME == RTVD )
__global__ void CUFLU_FluidSolver_RTVD(
   real g_Fluid_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
   real g_Fluid_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   real g_Flux     [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double g_Corner[][3],
   const real g_Pot_USG[][ CUBE(USG_NXT_F) ],
   const real dt, const real _dh, const real Gamma, const bool StoreFlux,
   const bool XYZ, const real MinDens, const real MinPres );
#elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
__global__
void CUFLU_FluidSolver_MHM(
   const real   Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
         char   DE_Array_Out [][ CUBE(PS2) ],
         real   Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double Corner_Array [][3],
   const real   Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   PriVar       [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   Slope_PPM    [][3][NCOMP_TOTAL][ CUBE(N_SLOPE_PPM) ],
         real   FC_Var       [][6][NCOMP_TOTAL][ CUBE(N_FC_VAR) ],
         real   FC_Flux      [][3][NCOMP_TOTAL][ CUBE(N_FC_FLUX) ],
   const real dt, const real dh, const real Gamma, const bool StoreFlux,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff,
   const double Time, const OptGravityType_t GravityType,
   const real MinDens, const real MinPres, const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool JeansMinPres, const real JeansMinPres_Coeff );
#if ( NCOMP_PASSIVE > 0 )
int CUFLU_SetConstMem_FluidSolver_NormIdx( int NormIdx_h[] );
#endif
#elif ( FLU_SCHEME == CTU )
__global__
void CUFLU_FluidSolver_CTU(
   const real   Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
         char   DE_Array_Out [][ CUBE(PS2) ],
         real   Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double Corner_Array [][3],
   const real   Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   PriVar       [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   Slope_PPM    [][3][NCOMP_TOTAL][ CUBE(N_SLOPE_PPM) ],
         real   FC_Var       [][6][NCOMP_TOTAL][ CUBE(N_FC_VAR) ],
         real   FC_Flux      [][3][NCOMP_TOTAL][ CUBE(N_FC_FLUX) ],
   const real dt, const real dh, const real Gamma, const bool StoreFlux,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff,
   const double Time, const OptGravityType_t GravityType,
   const real MinDens, const real MinPres, const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool JeansMinPres, const real JeansMinPres_Coeff );
#if ( NCOMP_PASSIVE > 0 )
int CUFLU_SetConstMem_FluidSolver_NormIdx( int NormIdx_h[] );
#endif
#endif // FLU_SCHEME
__global__ void CUFLU_dtSolver_HydroCFL( real g_dt_Array[], const real g_Flu_Array[][NCOMP_FLUID][ CUBE(PS1) ],
                                         const real dh, const real Safety, const real Gamma, const real MinPres );
#ifdef GRAVITY
__global__ void CUPOT_dtSolver_HydroGravity( real g_dt_Array[],
                                             const real g_Pot_Array[][ CUBE(GRA_NXT) ],
                                             const double g_Corner_Array[][3],
                                             const real dh, const real Safety, const bool P5_Gradient,
                                             const OptGravityType_t GravityType, const double ExtAcc_Time );
#endif
#elif ( MODEL == MHD )
#warning : WAIT MHD !!!

#elif ( MODEL == ELBDM )
__global__ void CUFLU_ELBDMSolver( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                   real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                   real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                                   const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                   const real Taylor3_Coeff, const bool XYZ, const real MinDens );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL


#ifdef GRAVITY

// Poisson solver prototypes
#if   ( POT_SCHEME == SOR )
#ifdef USE_PSOLVER_10TO14
__global__ void CUPOT_PoissonSolver_SOR_10to14cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ],
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ],
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6,
                                                    const real Const, const IntScheme_t IntScheme );
#else
__global__ void CUPOT_PoissonSolver_SOR_16to18cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ],
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ],
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6,
                                                    const real Const, const IntScheme_t IntScheme );
#endif // #ifdef USE_PSOLVER_10TO14 ... else ...
#elif ( POT_SCHEME == MG )
__global__ void CUPOT_PoissonSolver_MG( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ],
                                        const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ],
                                              real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                        const real dh_Min, const int Max_Iter, const int NPre_Smooth,
                                        const int NPost_Smooth, const real Tolerated_Error, const real Poi_Coeff,
                                        const IntScheme_t IntScheme );
#endif // POT_SCHEME


// Gravity solver prototypes in different models
#if   ( MODEL == HYDRO )
__global__
void CUPOT_HydroGravitySolver(
         real   Flu_Array_New[][GRA_NIN][ CUBE(PS1) ],
   const real   Pot_Array_New[][ CUBE(GRA_NXT) ],
   const double Corner_Array [][3],
   const real   Pot_Array_USG[][ CUBE(USG_NXT_G) ],
   const real   Flu_Array_USG[][GRA_NIN-1][ CUBE(PS1) ],
         char   DE_Array     [][ CUBE(PS1) ],
   const real dt, const real dh, const bool P5_Gradient,
   const OptGravityType_t GravityType,
   const double TimeNew, const double TimeOld, const real MinEint );

#elif ( MODEL == MHD )
#warning :: WAIT MHD !!!

#elif ( MODEL == ELBDM )
__global__ void CUPOT_ELBDMGravitySolver(       real g_Flu_Array[][GRA_NIN][ PS1*PS1*PS1 ],
                                          const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const double g_Corner_Array[][3],
                                          const real EtaDt, const real dh, const real Lambda, const bool ExtPot,
                                          const double Time );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

int CUPOT_SetConstMem_PoissonSolver();

#endif // GRAVITY




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Set_Default_GPU_Parameter
// Description :  Set several GPU parameters to the default values if they are not set in the input file
//
// Parameter   :  GPU_NStream     : Number of streams for the asynchronous memory copy in GPU
//                Flu_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the fluid solver
//                Pot_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the Poisson solver
//                Che_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the Grackle solver
//-------------------------------------------------------------------------------------------------------
void CUAPI_Set_Default_GPU_Parameter( int &GPU_NStream, int &Flu_GPU_NPGroup, int &Pot_GPU_NPGroup, int &Che_GPU_NPGroup )
{

   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ...\n", __FUNCTION__ );


// get the device ID
   int GetDeviceID = 999;
   CUDA_CHECK_ERROR(  hipGetDevice( &GetDeviceID )  );


// load the device properties
   hipDeviceProp_t DeviceProp;
   CUDA_CHECK_ERROR(  hipGetDeviceProperties( &DeviceProp, GetDeviceID )  );


// set the default GPU parameters
// (1) GPU_NSTREAM
   if ( GPU_NStream <= 0 )
   {
      if ( DeviceProp.deviceOverlap )
      {
#        if   ( MODEL == HYDRO )
#           if   ( GPU_ARCH == FERMI )
            GPU_NStream = 8;
#           elif ( GPU_ARCH == KEPLER )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == MAXWELL )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == PASCAL )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == VOLTA )
            GPU_NStream = 32;
#           else
#           error : UNKNOWN GPU_ARCH !!
#           endif

#        elif ( MODEL == MHD )
#        warning :: WAIT MHD !!!

#        elif ( MODEL == ELBDM )
#           if   ( GPU_ARCH == FERMI )
            GPU_NStream = 8;
#           elif ( GPU_ARCH == KEPLER )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == MAXWELL )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == PASCAL )
            GPU_NStream = 32;
#           elif ( GPU_ARCH == VOLTA )
            GPU_NStream = 32;
#           else
#           error : ERROR : UNKNOWN GPU_ARCH !!
#           endif
#        else
#           error : ERROR : UNKNOWN MODEL !!
#        endif // MODEL
      } // if ( DeviceProp.deviceOverlap )

      else
         GPU_NStream = 1;

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "GPU_NSTREAM", GPU_NSTREAM );
   } // if ( GPU_NStream <= 0 )


// (2) XXX_GPU_NPGROUP
// (2-1) FLU_GPU_NPGROUP
   if ( Flu_GPU_NPGroup <= 0 )
   {
#     if   ( MODEL == HYDRO )
#        if   ( GPU_ARCH == FERMI )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == KEPLER )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == MAXWELL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == PASCAL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == VOLTA )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
#        error : UNKNOWN GPU_ARCH !!
#        endif

#     elif ( MODEL == MHD )
#        warning :: WAIT MHD !!!

#     elif ( MODEL == ELBDM )
#        if   ( GPU_ARCH == FERMI )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == KEPLER )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == MAXWELL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == PASCAL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == VOLTA )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
#        error : UNKNOWN GPU_ARCH !!
#        endif
#     else
#        error : ERROR : UNKNOWN MODEL !!
#     endif // MODEL

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "FLU_GPU_NPGROUP", Flu_GPU_NPGroup );
   } // if ( Flu_GPU_NPGroup <= 0 )

// (2-2) POT_GPU_NPGROUP
#  ifdef GRAVITY
   if ( Pot_GPU_NPGroup <= 0 )
   {
#     if   ( GPU_ARCH == FERMI )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == KEPLER )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == MAXWELL )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == PASCAL )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == VOLTA )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     else
#     error : UNKNOWN GPU_ARCH !!
#     endif

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "POT_GPU_NPGROUP", Pot_GPU_NPGroup );
   } // if ( Pot_GPU_NPGroup <= 0 )
#  endif

// (2-3) CHE_GPU_NPGROUP
#  ifdef SUPPORT_GRACKLE
   if ( Che_GPU_NPGroup <= 0 )
   {
#     if   ( GPU_ARCH == FERMI )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == KEPLER )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == MAXWELL )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == PASCAL )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == VOLTA )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     else
#     error : UNKNOWN GPU_ARCH !!
#     endif

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "CHE_GPU_NPGROUP", Che_GPU_NPGroup );
   } // if ( Che_GPU_NPGroup <= 0 )
#  endif


// (3) cache preference
// (3-1) fluid solver
#  if   ( MODEL == HYDRO )
#  if   ( FLU_SCHEME == RTVD )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_RTVD),      hipFuncCachePreferShared )  );
#  elif ( FLU_SCHEME == MHM )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),       hipFuncCachePreferL1     )  );
#  elif ( FLU_SCHEME == MHM_RP )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),       hipFuncCachePreferL1     )  );
#  elif ( FLU_SCHEME == CTU )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_CTU),       hipFuncCachePreferL1     )  );
#  endif
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_dtSolver_HydroCFL),     hipFuncCachePreferShared )  );
#  ifdef GRAVITY
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_dtSolver_HydroGravity), hipFuncCachePreferShared )  );
#  endif

#  elif ( MODEL == MHD )
#  warning :: WAIT MHD !!!

#  elif ( MODEL == ELBDM )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_ELBDMSolver),      hipFuncCachePreferShared )  );

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL


#  ifdef GRAVITY

// (3-2) Poisson solver
#  if   ( POT_SCHEME == SOR )
#  ifdef USE_PSOLVER_10TO14
   CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_SOR_10to14cube), hipFuncCachePreferShared ) );
#  else
   CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_SOR_16to18cube), hipFuncCachePreferShared ) );
#  endif
#  elif ( POT_SCHEME == MG )
   CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_MG),             hipFuncCachePreferShared ) );
#  endif // POT_SCHEME


// (3-3) gravity solver
#  if   ( MODEL == HYDRO )
   CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_HydroGravitySolver),           hipFuncCachePreferShared ) );

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL == ELBDM )
   CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_ELBDMGravitySolver),           hipFuncCachePreferL1     ) );

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL

#  endif // GRAVITY


// (4) set the constant variables
// --> note that the auxiliary arrays for the external acceleration and potential are set by CUAPI_Init_ExternalAccPot()
#  if ( NCOMP_PASSIVE > 0 )
   if  ( OPT__NORMALIZE_PASSIVE )
   {
#     if ( MODEL == HYDRO  &&  ( FLU_SCHEME == MHM || FLU_SCHEME == MHM_RP || FLU_SCHEME == CTU )  )
      if ( CUFLU_SetConstMem_FluidSolver_NormIdx(PassiveNorm_VarIdx) != 0  )
         Aux_Error( ERROR_INFO, "CUFLU_SetConstMem_FluidSolver_NormIdx failed ...\n" );

#     elif ( MODEL == MHD )
#     warning : WAIT MHD !!!

#     endif // MODEL
   }
#  endif // #if ( NCOMP_PASSIVE > 0 )

#  ifdef GRAVITY
   if ( CUPOT_SetConstMem_PoissonSolver() != 0 )
      Aux_Error( ERROR_INFO, "CUPOT_SetConstMem_PoissonSolver failed ...\n" );
#  endif // #ifdef GRAVITY


   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ... done\n", __FUNCTION__ );

} // FUNCTION : CUAPI_Set_Default_GPU_Parameter



#endif // #ifdef GPU
