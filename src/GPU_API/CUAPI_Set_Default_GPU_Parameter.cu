#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "CUFLU.h"
#ifdef GRAVITY
#include "CUPOT.h"
#endif

#ifdef GPU



// fluid solver prototypes in different models
#if   ( MODEL == HYDRO )
#if   ( FLU_SCHEME == RTVD )
__global__ void CUFLU_FluidSolver_RTVD(
   real g_Fluid_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
   real g_Fluid_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   real g_Flux     [][9][NCOMP_TOTAL][ SQR(PS2) ],
   const double g_Corner[][3],
   const real g_Pot_USG[][ CUBE(USG_NXT_F) ],
   const real dt, const real _dh, const bool StoreFlux,
   const bool XYZ, const real MinDens, const real MinPres, const real MinEint,
   const EoS_t EoS );
#elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
__global__
void CUFLU_FluidSolver_MHM(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const int MinMod_MaxIter, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool FracPassive, const int NFrac,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_t EoS );
#elif ( FLU_SCHEME == CTU )
__global__
void CUFLU_FluidSolver_CTU(
   const real   g_Flu_Array_In [][NCOMP_TOTAL][ CUBE(FLU_NXT) ],
         real   g_Flu_Array_Out[][NCOMP_TOTAL][ CUBE(PS2) ],
   const real   g_Mag_Array_In [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_Mag_Array_Out[][NCOMP_MAG][ PS2P1*SQR(PS2) ],
         char   g_DE_Array_Out [][ CUBE(PS2) ],
         real   g_Flux_Array   [][9][NCOMP_TOTAL][ SQR(PS2) ],
         real   g_Ele_Array    [][9][NCOMP_ELE][ PS2P1*PS2 ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_F) ],
         real   g_PriVar       []   [NCOMP_LR            ][ CUBE(FLU_NXT) ],
         real   g_Slope_PPM    [][3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ],
         real   g_FC_Var       [][6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR) ],
         real   g_FC_Flux      [][3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX) ],
         real   g_FC_Mag_Half  [][NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ],
         real   g_EC_Ele       [][NCOMP_MAG][ CUBE(N_EC_ELE) ],
   const real dt, const real dh,
   const bool StoreFlux, const bool StoreElectric,
   const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, const double Time,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const real MinDens, const real MinPres, const real MinEint,
   const real DualEnergySwitch,
   const bool NormPassive, const int NNorm,
   const bool FracPassive, const int NFrac,
   const bool JeansMinPres, const real JeansMinPres_Coeff,
   const EoS_t EoS );
#endif // FLU_SCHEME
__global__ void CUFLU_dtSolver_HydroCFL( real g_dt_Array[], const real g_Flu_Array[][FLU_NIN_T][ CUBE(PS1) ],
                                         const real g_Mag_Array[][NCOMP_MAG][ PS1P1*SQR(PS1) ],
                                         const real dh, const real Safety, const real MinPres, const EoS_t EoS );
#ifdef GRAVITY
__global__
void CUPOT_dtSolver_HydroGravity( real g_dt_Array[], const real g_Pot_Array[][ CUBE(GRA_NXT) ],
                                  const double g_Corner_Array[][3],
                                  const real dh, const real Safety, const bool P5_Gradient,
                                  const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
                                  const double ExtAcc_Time );
#endif

#elif ( MODEL == ELBDM )
__global__ void CUFLU_ELBDMSolver( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                   real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                   real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                                   const real dt, const real _dh, const real Eta, const bool StoreFlux,
                                   const real Taylor3_Coeff, const bool XYZ, const real MinDens );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL


#ifdef GRAVITY

// Poisson solver prototypes
#if   ( POT_SCHEME == SOR )
__global__ void CUPOT_PoissonSolver_SOR( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ],
                                         const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ],
                                               real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                         const int Min_Iter, const int Max_Iter, const real Omega_6,
                                         const real Const, const IntScheme_t IntScheme );
#elif ( POT_SCHEME == MG )
__global__ void CUPOT_PoissonSolver_MG( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ],
                                        const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ],
                                              real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                        const real dh_Min, const int Max_Iter, const int NPre_Smooth,
                                        const int NPost_Smooth, const real Tolerated_Error, const real Poi_Coeff,
                                        const IntScheme_t IntScheme );
#endif // POT_SCHEME


// Gravity solver prototypes in different models
#if   ( MODEL == HYDRO )
__global__
void CUPOT_HydroGravitySolver(
         real   g_Flu_Array_New[][GRA_NIN][ CUBE(PS1) ],
   const real   g_Pot_Array_New[][ CUBE(GRA_NXT) ],
   const double g_Corner_Array [][3],
   const real   g_Pot_Array_USG[][ CUBE(USG_NXT_G) ],
   const real   g_Flu_Array_USG[][GRA_NIN-1][ CUBE(PS1) ],
         char   g_DE_Array     [][ CUBE(PS1) ],
   const real   g_Emag_Array   [][ CUBE(PS1) ],
   const real dt, const real dh, const bool P5_Gradient,
   const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
   const double TimeNew, const double TimeOld, const real MinEint );

#elif ( MODEL == ELBDM )
__global__
void CUPOT_ELBDMGravitySolver(       real g_Flu_Array[][GRA_NIN][ PS1*PS1*PS1 ],
                               const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                               const double g_Corner_Array[][3],
                               const real EtaDt, const real dh, const real Lambda );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

#endif // GRAVITY


// source-term solver prototype
__global__
void CUSRC_SrcSolver_IterateAllCells(
   const real g_Flu_Array_In [][FLU_NIN_S ][ CUBE(SRC_NXT)           ],
         real g_Flu_Array_Out[][FLU_NOUT_S][ CUBE(PS1)               ],
   const real g_Mag_Array_In [][NCOMP_MAG ][ SRC_NXT_P1*SQR(SRC_NXT) ],
   const double g_Corner_Array[][3],
   const SrcTerms_t SrcTerms, const int NPatchGroup, const real dt, const real dh,
   const double TimeNew, const double TimeOld,
   const real MinDens, const real MinPres, const real MinEint, const EoS_t EoS );




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Set_Default_GPU_Parameter
// Description :  Set several GPU parameters to the default values if they are not set in the input file
//
// Parameter   :  GPU_NStream     : Number of streams for the asynchronous memory copy in GPU
//                Flu_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the fluid solver
//                Pot_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the Poisson solver
//                Che_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the Grackle solver
//                Src_GPU_NPGroup : Number of patch groups sent into GPU simultaneously for the source-term solver
//-------------------------------------------------------------------------------------------------------
void CUAPI_Set_Default_GPU_Parameter( int &GPU_NStream, int &Flu_GPU_NPGroup, int &Pot_GPU_NPGroup, int &Che_GPU_NPGroup,
                                      int &Src_GPU_NPGroup )
{

   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ...\n", __FUNCTION__ );


// get the device ID
   int GetDeviceID = 999;
   CUDA_CHECK_ERROR(  hipGetDevice( &GetDeviceID )  );


// load the device properties
   hipDeviceProp_t DeviceProp;
   CUDA_CHECK_ERROR(  hipGetDeviceProperties( &DeviceProp, GetDeviceID )  );


// set the default GPU parameters
// (1) GPU_NSTREAM
   if ( GPU_NStream <= 0 )
   {
      if ( DeviceProp.deviceOverlap )
      {
#        if   ( MODEL == HYDRO )
#           if   ( GPU_ARCH == FERMI )
            GPU_NStream = 8;
#           elif ( GPU_ARCH == KEPLER )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == MAXWELL )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == PASCAL )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == VOLTA )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == TURING )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == AMPERE )
            GPU_NStream = 16;
#           else
#           error : UNKNOWN GPU_ARCH !!
#           endif

#        elif ( MODEL == ELBDM )
#           if   ( GPU_ARCH == FERMI )
            GPU_NStream = 8;
#           elif ( GPU_ARCH == KEPLER )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == MAXWELL )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == PASCAL )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == VOLTA )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == TURING )
            GPU_NStream = 16;
#           elif ( GPU_ARCH == AMPERE )
            GPU_NStream = 16;
#           else
#           error : ERROR : UNKNOWN GPU_ARCH !!
#           endif
#        else
#           error : ERROR : UNKNOWN MODEL !!
#        endif // MODEL
      } // if ( DeviceProp.deviceOverlap )

      else
         GPU_NStream = 1;

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "GPU_NSTREAM", GPU_NSTREAM );
   } // if ( GPU_NStream <= 0 )


// (2) XXX_GPU_NPGROUP
// (2-1) FLU_GPU_NPGROUP
   if ( Flu_GPU_NPGroup <= 0 )
   {
#     if   ( MODEL == HYDRO )
#        if   ( GPU_ARCH == FERMI )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == KEPLER )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == MAXWELL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == PASCAL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == VOLTA )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == TURING )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == AMPERE )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
#        error : UNKNOWN GPU_ARCH !!
#        endif

#     elif ( MODEL == ELBDM )
#        if   ( GPU_ARCH == FERMI )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == KEPLER )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == MAXWELL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == PASCAL )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == VOLTA )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == TURING )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        elif ( GPU_ARCH == AMPERE )
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
#        error : UNKNOWN GPU_ARCH !!
#        endif
#     else
#        error : ERROR : UNKNOWN MODEL !!
#     endif // MODEL

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "FLU_GPU_NPGROUP", Flu_GPU_NPGroup );
   } // if ( Flu_GPU_NPGroup <= 0 )

// (2-2) POT_GPU_NPGROUP
#  ifdef GRAVITY
   if ( Pot_GPU_NPGroup <= 0 )
   {
#     if   ( GPU_ARCH == FERMI )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == KEPLER )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == MAXWELL )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == PASCAL )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == VOLTA )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == TURING )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == AMPERE )
      Pot_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     else
#     error : UNKNOWN GPU_ARCH !!
#     endif

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "POT_GPU_NPGROUP", Pot_GPU_NPGroup );
   } // if ( Pot_GPU_NPGroup <= 0 )
#  endif

// (2-3) CHE_GPU_NPGROUP
#  ifdef SUPPORT_GRACKLE
   if ( Che_GPU_NPGroup <= 0 )
   {
#     if   ( GPU_ARCH == FERMI )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == KEPLER )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == MAXWELL )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == PASCAL )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == VOLTA )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == TURING )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == AMPERE )
      Che_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     else
#     error : UNKNOWN GPU_ARCH !!
#     endif

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "CHE_GPU_NPGROUP", Che_GPU_NPGroup );
   } // if ( Che_GPU_NPGroup <= 0 )
#  endif

// (2-4) SRC_GPU_NPGROUP
   if ( Src_GPU_NPGroup <= 0 )
   {
#     if   ( GPU_ARCH == FERMI )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == KEPLER )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == MAXWELL )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == PASCAL )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == VOLTA )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == TURING )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     elif ( GPU_ARCH == AMPERE )
      Src_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#     else
#     error : UNKNOWN GPU_ARCH !!
#     endif

      if ( MPI_Rank == 0 )
         Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d"
                              " --> might be further fine-tuned\n", "SRC_GPU_NPGROUP", Src_GPU_NPGroup );
   } // if ( Src_GPU_NPGroup <= 0 )


// (3) cache preference
// (3-1) fluid solver
#  if   ( MODEL == HYDRO )
#  if   ( FLU_SCHEME == RTVD )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_RTVD),             hipFuncCachePreferShared )  );
#  elif ( FLU_SCHEME == MHM )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),              hipFuncCachePreferL1     )  );
#  elif ( FLU_SCHEME == MHM_RP )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),              hipFuncCachePreferL1     )  );
#  elif ( FLU_SCHEME == CTU )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_CTU),              hipFuncCachePreferL1     )  );
#  endif
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_dtSolver_HydroCFL),            hipFuncCachePreferShared )  );
#  ifdef GRAVITY
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_dtSolver_HydroGravity),        hipFuncCachePreferShared )  );
#  endif

#  elif ( MODEL == ELBDM )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_ELBDMSolver),                  hipFuncCachePreferShared )  );

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL


#  ifdef GRAVITY

// (3-2) Poisson solver
#  if   ( POT_SCHEME == SOR )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_SOR),            hipFuncCachePreferShared )  );
#  elif ( POT_SCHEME == MG )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_MG),             hipFuncCachePreferShared )  );
#  endif // POT_SCHEME


// (3-3) gravity solver
#  if   ( MODEL == HYDRO )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_HydroGravitySolver),           hipFuncCachePreferShared )  );

#  elif ( MODEL == ELBDM )
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_ELBDMGravitySolver),           hipFuncCachePreferL1     )  );

#  else
#  error : ERROR : unsupported MODEL !!
#  endif // MODEL

#  endif // GRAVITY


// (3-4) source-term solver
   CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUSRC_SrcSolver_IterateAllCells),   hipFuncCachePreferL1      )  );


   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ... done\n", __FUNCTION__ );

} // FUNCTION : CUAPI_Set_Default_GPU_Parameter



#endif // #ifdef GPU
