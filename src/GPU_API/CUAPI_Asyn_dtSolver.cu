#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "CUFLU.h"
#ifdef GRAVITY
#include "CUPOT.h"
#endif

#ifdef GPU



#if   ( MODEL == HYDRO )
__global__
void CUFLU_dtSolver_HydroCFL( real g_dt_Array[], const real g_Flu_Array[][FLU_NIN_T][ CUBE(PS1) ],
                              const real g_Mag_Array[][NCOMP_MAG][ PS1P1*SQR(PS1) ],
                              const real dh, const real Safety, const real MinPres,
                              const EoS_DE2P_t EoS_DensEint2Pres_Func, const EoS_DP2C_t EoS_DensPres2CSqr_Func,
                              const EoS_TEM2C_t EoS_Temper2CSqr_Func, const EoS_GUESS_t EoS_GuessHTilde_Func,
                              const EoS_H2TEM_t EoS_HTilde2Temp_Func );
#ifdef GRAVITY
__global__
void CUPOT_dtSolver_HydroGravity( real g_dt_Array[], const real g_Pot_Array[][ CUBE(GRA_NXT) ],
                                  const double g_Corner_Array[][3],
                                  const real dh, const real Safety, const bool P5_Gradient,
                                  const bool UsePot, const OptExtAcc_t ExtAcc, const ExtAcc_t ExtAcc_Func,
                                  const double ExtAcc_Time );
#endif

#elif ( MODEL == ELBDM )

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

// device pointers
extern real *d_dt_Array_T;
extern real (*d_Flu_Array_T)[FLU_NIN_T][ CUBE(PS1) ];
#ifdef GRAVITY
extern real (*d_Pot_Array_T)[ CUBE(GRA_NXT) ];
extern double (*d_Corner_Array_PGT)[3];
#endif
#ifdef MHD
extern real (*d_Mag_Array_T)[NCOMP_MAG][ PS1P1*SQR(PS1) ];
#else
static real (*d_Mag_Array_T)[NCOMP_MAG][ PS1P1*SQR(PS1) ] = NULL;
#endif

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Asyn_dtSolver
// Description :  Invoke various dt solvers
//
//                ***********************************************************
//                **                Asynchronous Function                  **
//                **                                                       **
//                **  will return before the execution in GPU is complete  **
//                ***********************************************************
//
// Note        :  1. Use streams for the asychronous memory copy between device and host
//                2. Prefix "d" : for pointers pointing to the "Device" memory space
//                   Prefix "h" : for pointers pointing to the "Host"   memory space
//
// Parameter   :  TSolver        : Target dt solver
//                                 --> DT_FLU_SOLVER : dt solver for fluid
//                                     DT_GRA_SOLVER : dt solver for gravity
//                h_dt_Array     : Host array to store the minimum dt in each target patch
//                h_Flu_Array    : Host array storing the prepared fluid     data of each target patch
//                h_Mag_Array    : Host array storing the prepared B field   data of each target patch
//                h_Pot_Array    : Host array storing the prepared potential data of each target patch
//                h_Corner_Array : Array storing the physical corner coordinates of each patch
//                NPatchGroup    : Number of patch groups evaluated simultaneously by GPU
//                dh             : Grid size
//                Safety         : dt safety factor
//                MinPres        : Minimum allowed pressure
//                P5_Gradient    : Use 5-points stencil to evaluate the potential gradient
//                UsePot         : Add self-gravity and/or external potential
//                ExtAcc         : Add external acceleration
//                TargetTime     : Target physical time
//                GPU_NStream    : Number of CUDA streams for the asynchronous memory copy
//
// Return      :  h_dt_Array
//-------------------------------------------------------------------------------------------------------
void CUAPI_Asyn_dtSolver( const Solver_t TSolver, real h_dt_Array[], const real h_Flu_Array[][FLU_NIN_T][ CUBE(PS1) ],
                          const real h_Mag_Array[][NCOMP_MAG][ PS1P1*SQR(PS1) ], const real h_Pot_Array[][ CUBE(GRA_NXT) ],
                          const double h_Corner_Array[][3], const int NPatchGroup, const real dh, const real Safety,
                          const real MinPres, const bool P5_Gradient, const bool UsePot, const OptExtAcc_t ExtAcc,
                          const double TargetTime, const int GPU_NStream )
{

// check
#  ifdef GAMER_DEBUG
   if ( TSolver != DT_FLU_SOLVER )
#  ifdef GRAVITY
   if ( TSolver != DT_GRA_SOLVER )
#  endif
      Aux_Error( ERROR_INFO, "TSolver != DT_FLU_SOLVER / DT_GRA_SOLVER !!\n" );

   if ( h_dt_Array == NULL )
      Aux_Error( ERROR_INFO, "h_dt_Array == NULL !!\n" );

   if ( TSolver == DT_FLU_SOLVER  &&  h_Flu_Array == NULL )
      Aux_Error( ERROR_INFO, "h_Flu_Array == NULL !!\n" );

#  ifdef GRAVITY
   if ( TSolver == DT_GRA_SOLVER )
   {
      if ( UsePot  &&  h_Pot_Array == NULL )
         Aux_Error( ERROR_INFO, "h_Pot_Array == NULL !!\n" );

      if ( ExtAcc )
      {
         if ( h_Corner_Array     == NULL )   Aux_Error( ERROR_INFO, "h_Corner_Array == NULL !!\n" );
         if ( d_Corner_Array_PGT == NULL )   Aux_Error( ERROR_INFO, "d_Corner_Array_PGT == NULL !!\n" );
      }
   }
#  endif

#  ifdef MHD
   if ( TSolver == DT_FLU_SOLVER  &&  h_Mag_Array == NULL )
      Aux_Error( ERROR_INFO, "h_Mag_Array == NULL !!\n" );
#  endif
#  endif // #ifdef GAMER_DEBUG


// set the block size
   const int NPatch = NPatchGroup*8;
   dim3 BlockDim_dtSolver( 1, 1, 1 );

   switch ( TSolver )
   {
      case DT_FLU_SOLVER:
         BlockDim_dtSolver.x = DT_FLU_BLOCK_SIZE;
      break;

#     ifdef GRAVITY
      case DT_GRA_SOLVER:
         BlockDim_dtSolver.x = DT_GRA_BLOCK_SIZE;
      break;
#     endif

      default :
         Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "TSolver", TSolver );
   }


// set the number of patches and the corresponding data size to be transferred into GPU in each stream
   int *NPatch_per_Stream = new int [GPU_NStream];
   int *UsedPatch         = new int [GPU_NStream];
   int *dt_MemSize        = new int [GPU_NStream];
   int *Corner_MemSize    = new int [GPU_NStream];

   int *Flu_MemSize       = ( TSolver == DT_FLU_SOLVER ) ? new int [GPU_NStream] : NULL;
#  ifdef MHD
   int *Mag_MemSize       = ( TSolver == DT_FLU_SOLVER ) ? new int [GPU_NStream] : NULL;
#  endif
#  ifdef GRAVITY
   int *Pot_MemSize       = ( TSolver == DT_GRA_SOLVER ) ? new int [GPU_NStream] : NULL;
#  endif


// number of patches in each stream
   UsedPatch[0] = 0;

   if ( GPU_NStream == 1 )    NPatch_per_Stream[0] = NPatch;
   else
   {
      for (int s=0; s<GPU_NStream-1; s++)
      {
         NPatch_per_Stream[s] = NPatch / GPU_NStream;
         UsedPatch[s+1] = UsedPatch[s] + NPatch_per_Stream[s];
      }

      NPatch_per_Stream[GPU_NStream-1] = NPatch - UsedPatch[GPU_NStream-1];
   }

// corresponding data size to be transferred into GPU in each stream
   for (int s=0; s<GPU_NStream; s++)
   {
      switch ( TSolver )
      {
         case DT_FLU_SOLVER:
            Flu_MemSize   [s] = sizeof(real  )*NPatch_per_Stream[s]*CUBE(PS1)*FLU_NIN_T;
#           ifdef MHD
            Mag_MemSize   [s] = sizeof(real  )*NPatch_per_Stream[s]*PS1P1*SQR(PS1)*NCOMP_MAG;
#           endif
         break;

#        ifdef GRAVITY
         case DT_GRA_SOLVER:
            Pot_MemSize   [s] = sizeof(real  )*NPatch_per_Stream[s]*CUBE(GRA_NXT);
            Corner_MemSize[s] = sizeof(double)*NPatch_per_Stream[s]*3;
         break;
#        endif

         default :
            Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "TSolver", TSolver );
      }

      dt_MemSize[s] = sizeof(real)*NPatch_per_Stream[s];
   }


// a. copy data from host to device
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      switch ( TSolver )
      {
         case DT_FLU_SOLVER:
            CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Flu_Array_T      + UsedPatch[s], h_Flu_Array    + UsedPatch[s],
                               Flu_MemSize[s],    hipMemcpyHostToDevice, Stream[s] )  );
#           ifdef MHD
            CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Mag_Array_T      + UsedPatch[s], h_Mag_Array    + UsedPatch[s],
                               Mag_MemSize[s],    hipMemcpyHostToDevice, Stream[s] )  );
#           endif
         break;

#        ifdef GRAVITY
         case DT_GRA_SOLVER:
            if ( UsePot )
            CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Pot_Array_T      + UsedPatch[s], h_Pot_Array    + UsedPatch[s],
                               Pot_MemSize[s],    hipMemcpyHostToDevice, Stream[s] )  );

            if ( ExtAcc )
            CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Corner_Array_PGT + UsedPatch[s], h_Corner_Array + UsedPatch[s],
                               Corner_MemSize[s], hipMemcpyHostToDevice, Stream[s] )  );
         break;
#        endif

         default :
            Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "TSolver", TSolver );
      }
   } // for (int s=0; s<GPU_NStream; s++)


// b. execute the kernel
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

#     if   ( MODEL == HYDRO )
      switch ( TSolver )
      {
         case DT_FLU_SOLVER:
            CUFLU_dtSolver_HydroCFL <<< NPatch_per_Stream[s], BlockDim_dtSolver, 0, Stream[s] >>>
                                    ( d_dt_Array_T  + UsedPatch[s],
                                      d_Flu_Array_T + UsedPatch[s],
                                      d_Mag_Array_T + UsedPatch[s],
                                      dh, Safety, MinPres,
                                      EoS_DensEint2Pres_GPUPtr, EoS_DensPres2CSqr_GPUPtr, EoS_Temper2CSqr_GPUPtr,
                                      EoS_GuessHTilde_GPUPtr, EoS_HTilde2Temp_GPUPtr );
         break;

#        ifdef GRAVITY
         case DT_GRA_SOLVER:
            CUPOT_dtSolver_HydroGravity <<< NPatch_per_Stream[s], BlockDim_dtSolver, 0, Stream[s] >>>
                                        ( d_dt_Array_T       + UsedPatch[s],
                                          d_Pot_Array_T      + UsedPatch[s],
                                          d_Corner_Array_PGT + UsedPatch[s],
                                          dh, Safety, P5_Gradient, UsePot, ExtAcc, GPUExtAcc_Ptr, TargetTime );
         break;
#        endif

         default :
            Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "TSolver", TSolver );
      }

#     elif ( MODEL == ELBDM )

#     else
#        error : unsupported MODEL !!
#     endif // MODEL

      CUDA_CHECK_ERROR( hipGetLastError() );
   } // for (int s=0; s<GPU_NStream; s++)


// c. copy data from device to host
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      CUDA_CHECK_ERROR(  hipMemcpyAsync( h_dt_Array + UsedPatch[s], d_dt_Array_T + UsedPatch[s],
                         dt_MemSize[s], hipMemcpyDeviceToHost, Stream[s] )  );
   } // for (int s=0; s<GPU_NStream; s++)


   delete [] NPatch_per_Stream;
   delete [] UsedPatch;
   delete [] dt_MemSize;
   delete [] Corner_MemSize;
   delete [] Flu_MemSize;
#  ifdef MHD
   delete [] Mag_MemSize;
#  endif
#  ifdef GRAVITY
   delete [] Pot_MemSize;
#  endif

} // FUNCTION : CUAPI_Asyn_dtSolver



#endif // #ifdef GPU
