#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU



extern real (*d_Flu_Array_F_In )[FLU_NIN ][ CUBE(FLU_NXT) ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ CUBE(PS2) ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ SQR(PS2) ];
#ifdef UNSPLIT_GRAVITY
extern double (*d_Corner_Array_F)[3];
#endif
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ CUBE(PS2) ];
#endif
extern real *d_dt_Array_T;
extern real (*d_Flu_Array_T)[NCOMP_FLUID][ CUBE(PS1) ];

// global memory arrays in different models
#if   ( MODEL == HYDRO )
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)      [NCOMP_TOTAL][ CUBE(FLU_NXT)     ];
extern real (*d_Slope_PPM)[3][NCOMP_TOTAL][ CUBE(N_SLOPE_PPM) ];
extern real (*d_FC_Var)   [6][NCOMP_TOTAL][ CUBE(N_FC_VAR)    ];
extern real (*d_FC_Flux)  [3][NCOMP_TOTAL][ CUBE(N_FC_FLUX)   ];
#endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#elif ( MODEL == MHD )
#warning : WAIT MHD !!!

#elif ( MODEL != ELBDM )
#warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#endif // MODEL

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_Fluid
// Description :  Free the GPU and CPU memory previously allocated by CUAPI_MemAllocate_Fluid()
//
// Parameter   :  GPU_NStream : Number of CUDA streams for the asynchronous memory copy
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_Fluid( const int GPU_NStream )
{

// free the device memory (used by all models)
   if ( d_Flu_Array_F_In  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_In  )  );  d_Flu_Array_F_In  = NULL; }
   if ( d_Flu_Array_F_Out != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_Out )  );  d_Flu_Array_F_Out = NULL; }
   if ( d_Flux_Array      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flux_Array      )  );  d_Flux_Array      = NULL; }
#  ifdef UNSPLIT_GRAVITY
   if ( d_Corner_Array_F  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Corner_Array_F  )  );  d_Corner_Array_F  = NULL; }
#  endif
#  ifdef DUAL_ENERGY
   if ( d_DE_Array_F_Out  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_DE_Array_F_Out  )  );  d_DE_Array_F_Out  = NULL; }
#  endif
   if ( d_dt_Array_T      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_dt_Array_T      )  );  d_dt_Array_T      = NULL; }
   if ( d_Flu_Array_T     != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_T     )  );  d_Flu_Array_T     = NULL; }


// free the device memory (used by different models)
#  if   ( MODEL == HYDRO )
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   if ( d_PriVar    != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_PriVar    )  );  d_PriVar    = NULL; }
   if ( d_Slope_PPM != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Slope_PPM )  );  d_Slope_PPM = NULL; }
   if ( d_FC_Var    != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_FC_Var    )  );  d_FC_Var    = NULL; }
   if ( d_FC_Flux   != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_FC_Flux   )  );  d_FC_Flux   = NULL; }
#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_Flu_Array_F_In [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_In [t] )  );  h_Flu_Array_F_In [t] = NULL; }
      if ( h_Flu_Array_F_Out[t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_Out[t] )  );  h_Flu_Array_F_Out[t] = NULL; }
      if ( h_Flux_Array     [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flux_Array     [t] )  );  h_Flux_Array     [t] = NULL; }
#     ifdef UNSPLIT_GRAVITY
      if ( h_Corner_Array_F [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Corner_Array_F [t] )  );  h_Corner_Array_F [t] = NULL; }
#     endif
#     ifdef DUAL_ENERGY
      if ( h_DE_Array_F_Out [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_DE_Array_F_Out [t] )  );  h_DE_Array_F_Out [t] = NULL; }
#     endif
      if ( h_dt_Array_T     [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_dt_Array_T     [t] )  );  h_dt_Array_T     [t] = NULL; }
      if ( h_Flu_Array_T    [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_T    [t] )  );  h_Flu_Array_T    [t] = NULL; }
   } // for (int t=0; t<2; t++)


// destroy streams
   if ( Stream != NULL )
   {
      for (int s=0; s<GPU_NStream; s++)   CUDA_CHECK_ERROR(  hipStreamDestroy( Stream[s] )  );

      delete [] Stream;
      Stream = NULL;
   }

} // FUNCTION : CUAPI_MemFree_Fluid



#endif // #ifdef GPU
