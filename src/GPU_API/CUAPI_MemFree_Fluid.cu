#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU



extern real (*d_Flu_Array_F_In )[FLU_NIN ][ CUBE(FLU_NXT) ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ CUBE(PS2) ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ SQR(PS2) ];
#ifdef UNSPLIT_GRAVITY
extern double (*d_Corner_Array_F)[3];
#endif
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ CUBE(PS2) ];
#endif
#ifdef MHD
extern real (*d_Mag_Array_F_In )[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_Mag_Array_F_Out)[NCOMP_MAG][ PS2P1*SQR(PS2)          ];
extern real (*d_Ele_Array      )[9][NCOMP_ELE][ PS2P1*PS2 ];
extern real (*d_Mag_Array_T)[NCOMP_MAG][ PS1P1*SQR(PS1) ];
#endif
extern real *d_dt_Array_T;
extern real (*d_Flu_Array_T)[FLU_NIN_T][ CUBE(PS1) ];
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)      [NCOMP_LR            ][ CUBE(FLU_NXT)     ];
extern real (*d_Slope_PPM)[3][NCOMP_LR            ][ CUBE(N_SLOPE_PPM) ];
extern real (*d_FC_Var)   [6][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_VAR)    ];
extern real (*d_FC_Flux)  [3][NCOMP_TOTAL_PLUS_MAG][ CUBE(N_FC_FLUX)   ];
#ifdef MHD
extern real (*d_FC_Mag_Half)[NCOMP_MAG][ FLU_NXT_P1*SQR(FLU_NXT) ];
extern real (*d_EC_Ele     )[NCOMP_MAG][ CUBE(N_EC_ELE)          ];
#endif
#endif // FLU_SCHEME

#if ( MODEL != HYDRO  &&  MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#endif

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_Fluid
// Description :  Free the GPU and CPU memory previously allocated by CUAPI_MemAllocate_Fluid()
//
// Parameter   :  GPU_NStream : Number of CUDA streams for the asynchronous memory copy
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_Fluid( const int GPU_NStream )
{

// free the device memory
   if ( d_Flu_Array_F_In  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_In  )  );  d_Flu_Array_F_In  = NULL; }
   if ( d_Flu_Array_F_Out != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_Out )  );  d_Flu_Array_F_Out = NULL; }
   if ( d_Flux_Array      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flux_Array      )  );  d_Flux_Array      = NULL; }
#  ifdef UNSPLIT_GRAVITY
   if ( d_Corner_Array_F  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Corner_Array_F  )  );  d_Corner_Array_F  = NULL; }
#  endif
#  ifdef DUAL_ENERGY
   if ( d_DE_Array_F_Out  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_DE_Array_F_Out  )  );  d_DE_Array_F_Out  = NULL; }
#  endif
#  ifdef MHD
   if ( d_Mag_Array_F_In  != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Mag_Array_F_In  )  );  d_Mag_Array_F_In  = NULL; }
   if ( d_Mag_Array_F_Out != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Mag_Array_F_Out )  );  d_Mag_Array_F_Out = NULL; }
   if ( d_Ele_Array       != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Ele_Array       )  );  d_Ele_Array       = NULL; }
   if ( d_Mag_Array_T     != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Mag_Array_T     )  );  d_Mag_Array_T     = NULL; }
#  endif
   if ( d_dt_Array_T      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_dt_Array_T      )  );  d_dt_Array_T      = NULL; }
   if ( d_Flu_Array_T     != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_T     )  );  d_Flu_Array_T     = NULL; }
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   if ( d_PriVar      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_PriVar      )  );  d_PriVar      = NULL; }
   if ( d_Slope_PPM   != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_Slope_PPM   )  );  d_Slope_PPM   = NULL; }
   if ( d_FC_Var      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_FC_Var      )  );  d_FC_Var      = NULL; }
   if ( d_FC_Flux     != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_FC_Flux     )  );  d_FC_Flux     = NULL; }
#  ifdef MHD
   if ( d_FC_Mag_Half != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_FC_Mag_Half )  );  d_FC_Mag_Half = NULL; }
   if ( d_EC_Ele      != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_EC_Ele      )  );  d_EC_Ele      = NULL; }
#  endif
#  endif // FLU_SCHEME

#  if ( MODEL != HYDRO  &&  MODEL != ELBDM )
#    warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_Flu_Array_F_In [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_In [t] )  );  h_Flu_Array_F_In [t] = NULL; }
      if ( h_Flu_Array_F_Out[t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_Out[t] )  );  h_Flu_Array_F_Out[t] = NULL; }
      if ( h_Flux_Array     [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flux_Array     [t] )  );  h_Flux_Array     [t] = NULL; }
#     ifdef UNSPLIT_GRAVITY
      if ( h_Corner_Array_F [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Corner_Array_F [t] )  );  h_Corner_Array_F [t] = NULL; }
#     endif
#     ifdef DUAL_ENERGY
      if ( h_DE_Array_F_Out [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_DE_Array_F_Out [t] )  );  h_DE_Array_F_Out [t] = NULL; }
#     endif
#     ifdef MHD
      if ( h_Mag_Array_F_In [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Mag_Array_F_In [t] )  );  h_Mag_Array_F_In [t] = NULL; }
      if ( h_Mag_Array_F_Out[t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Mag_Array_F_Out[t] )  );  h_Mag_Array_F_Out[t] = NULL; }
      if ( h_Ele_Array      [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Ele_Array      [t] )  );  h_Ele_Array      [t] = NULL; }
      if ( h_Mag_Array_T    [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Mag_Array_T    [t] )  );  h_Mag_Array_T    [t] = NULL; }
#     endif
      if ( h_dt_Array_T     [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_dt_Array_T     [t] )  );  h_dt_Array_T     [t] = NULL; }
      if ( h_Flu_Array_T    [t] != NULL ) {  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_T    [t] )  );  h_Flu_Array_T    [t] = NULL; }
   } // for (int t=0; t<2; t++)


// destroy streams
   if ( Stream != NULL )
   {
      for (int s=0; s<GPU_NStream; s++)   CUDA_CHECK_ERROR(  hipStreamDestroy( Stream[s] )  );

      delete [] Stream;
      Stream = NULL;
   }

} // FUNCTION : CUAPI_MemFree_Fluid



#endif // #ifdef GPU
